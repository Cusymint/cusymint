#include "hip/hip_runtime.h"
#include "IntegralCommons.cuh"

#define HEURISTIC_TEST(_name, _integral, _expected_result) \
    INTEGRAL_TEST(Heuristic, _name, _integral, _expected_result)

namespace Test {
    class Heuristic : public IntegrationFixture {};

    HEURISTIC_TEST(EToX, "e^x*cos(e^x)", "sin(e^x)")
    HEURISTIC_TEST(EToXTower, "e^x*e^e^x*e^e^e^x*e^e^e^e^x*e^e^e^e^e^x*e^e^e^e^e^e^x",
                   "e^e^e^e^e^e^x")

    HEURISTIC_TEST(IntegralWithConstant, "10/(1+x^2)", "arctan(x)*10")
    HEURISTIC_TEST(Polynomial, "x^3+3*x^2+8*x+12", "(1/4)*(x^4)+x^3+(x^2)*4+x*12")
    HEURISTIC_TEST(ArbitraryProduct, "pi*2*e*e^x*10*sin(e^x)*ln(pi)", "-1*cos(e^x)*ln(pi)*20*e*pi")

    HEURISTIC_TEST(SumIntegral, "cos(x)+sin(x)", "sin(x)-cos(x)")
    HEURISTIC_TEST(LongSumIntegral, "1+cos(x)+sin(x)+1/(1+x^2)+pi+e",
                   "sin(x)-cos(x)+arctan(x)+x*(pi+e+1)")

    HEURISTIC_TEST(SineSubstitution, "cos(x)*e^sin(x)", "e^sin(x)")
    HEURISTIC_TEST(CosineSubstitution, "5cos^4(x)sin(x)", "-1*cos^5(x)")

    // TODO: Universal substitution when simplification is powerful enough
};

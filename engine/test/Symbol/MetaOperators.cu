#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include <string>
#include <vector>

#include "Evaluation/Integrate.cuh"
#include "Parser/Parser.cuh"
#include "Symbol/MetaOperators.cuh"
#include "Symbol/Product.cuh"
#include "Symbol/Symbol.cuh"
#include "Symbol/Variable.cuh"

// This is a workaround for use of commas in template types in macros
template <class T> struct macro_type;
template <class T, class U> struct macro_type<T(U)> {
    using type = U;
};
#define MACRO_TYPE(_pattern) macro_type<void(_pattern)>::type

#define _META_TEST_MATCH(_name, _pattern, _expression, _should_match)      \
    TEST(MetaOperatorsMatchTest, _name) { test_meta_match<MACRO_TYPE(_pattern), _should_match>(_expression); } // NOLINT

#define META_TEST_MATCH(_name, _pattern, _expression) \
    _META_TEST_MATCH(_name, _pattern, _expression, true)

#define META_TEST_NOT_MATCH(_name, _pattern, _expression) \
    _META_TEST_MATCH(_name, _pattern, _expression, false)

#define _META_TEST_MATCH_PAIR(_name, _pattern1, _pattern2, _expression1, _expression2, _should_match) \
    TEST(MetaOperatorsMatchTest, _name) { test_meta_match_pair<MACRO_TYPE(_pattern1), MACRO_TYPE(_pattern2), _should_match>(_expression1, _expression2); } // NOLINT

#define META_TEST_MATCH_PAIR(_name, _pattern1, _pattern2, _expression1, _expression2) \
    _META_TEST_MATCH_PAIR(_name, _pattern1, _pattern2, _expression1, _expression2, true)

#define META_TEST_NOT_MATCH_PAIR(_name, _pattern1, _pattern2, _expression1, _expression2) \
    _META_TEST_MATCH_PAIR(_name, _pattern1, _pattern2, _expression1, _expression2, false)

#define META_TEST_INIT(_name, _pattern, ...)               \
    TEST(MetaOperatorsInitTest, _name) { test_meta_init<MACRO_TYPE(_pattern)>(__VA_ARGS__); } // NOLINT

namespace Test {
    namespace {
        template <class T, bool SHOULD_MATCH>
        void test_meta_match(const std::vector<Sym::Symbol>& expression) {
            if constexpr (SHOULD_MATCH) {
                EXPECT_TRUE(T::match(*expression.data()));
            }
            else {
                EXPECT_FALSE(T::match(*expression.data()));
            }
        }

        template <class T, bool SHOULD_MATCH> void test_meta_match(const std::string& expression) {
            test_meta_match<T, SHOULD_MATCH>(Parser::parse_function(expression));
        }

        template <class T1, class T2, bool SHOULD_MATCH>
        void test_meta_match_pair(const std::vector<Sym::Symbol>& expression1, const std::vector<Sym::Symbol>& expression2) {
            if constexpr (SHOULD_MATCH) {
                EXPECT_TRUE(MACRO_TYPE((Sym::PatternPair<T1, T2>))::match_pair(*expression1.data(), *expression2.data()));
            }
            else {
                EXPECT_FALSE(MACRO_TYPE((Sym::PatternPair<T1, T2>))::match_pair(*expression1.data(), *expression2.data()));
            }
        }

        template <class T1, class T2, bool SHOULD_MATCH>
        void test_meta_match_pair(const std::string& expression1, const std::string& expression2) {
            test_meta_match_pair<T1, T2, SHOULD_MATCH>(Parser::parse_function(expression1), Parser::parse_function(expression2));
        }

        template <class T, class... Args>
        void test_meta_init(const std::vector<Sym::Symbol>& expected_expression,
                            const Args&... args) {
            std::vector<Sym::Symbol> expression(Sym::EXPRESSION_MAX_SYMBOL_COUNT);
            T::init(*expression.data(), {args...});
            expression.resize(expression[0].size());
            EXPECT_TRUE(Sym::Symbol::compare_trees(expression.data(), expected_expression.data()))
                << "Expressions do not match:\n"
                << expression.data()->to_string() << " <- got,\n"
                << expected_expression.data()->to_string() << " <- expected\n";
        }

        template <class T, class... Args>
        void test_meta_init(const std::string& expected_expression, const Args&... args) {
            test_meta_init<T, Args...>(Parser::parse_function(expected_expression), args...);
        }
    }

    META_TEST_INIT(Variable, Sym::Var, "x")
    META_TEST_INIT(Pi, Sym::Pi, "pi")
    META_TEST_INIT(E, Sym::E, "e")
    META_TEST_INIT(Integer, Sym::Integer<69>, "69")
    META_TEST_INIT(NumericConstant, Sym::Num, "123.456", 123.456)
    META_TEST_INIT(Copy, Sym::Copy, "x^2", *(Sym::var() ^ Sym::num(2)).data())
    // Simple OneArgOperators
    META_TEST_INIT(Sine, Sym::Sin<Sym::E>, "sin(e)")
    META_TEST_INIT(Cosine, Sym::Cos<Sym::Var>, "cos(x)")
    META_TEST_INIT(Tangent, Sym::Tan<Sym::Pi>, "tan(pi)")
    META_TEST_INIT(Cotangent, Sym::Cot<Sym::Sin<Sym::Var>>, "cot(sin(x))")
    META_TEST_INIT(Arcsine, Sym::Arcsin<Sym::E>, "arcsin(e)")
    META_TEST_INIT(Arccosine, Sym::Arccos<Sym::E>, "arccos(e)")
    META_TEST_INIT(Arctangent, Sym::Arctan<Sym::E>, "arctan(e)")
    META_TEST_INIT(Arccotangent, Sym::Arccot<Sym::E>, "arccot(e)")
    META_TEST_INIT(Logarithm, Sym::Ln<Sym::Var>, "ln(x)")
    // Simple TwoArgOperators
    META_TEST_INIT(Sum, (Sym::Add<Sym::Cos<Sym::E>, Sym::Pi>), "cos(e)+pi")
    META_TEST_INIT(Product, (Sym::Mul<Sym::Cos<Sym::Var>, Sym::Pi>), "cos(x)*pi")
    META_TEST_INIT(Power, (Sym::Pow<Sym::Cos<Sym::E>, Sym::Pi>), "cos(e)^pi")
    // Advanced expressions
    META_TEST_INIT(LongSum, (Sym::Sum<Sym::Var, Sym::Cos<Sym::Ln<Sym::Mul<Sym::Num, Sym::Var>>>, Sym::E, Sym::Integer<1>>), "x+(cos(ln(2*x))+(e+1))", 2)
    META_TEST_INIT(LongProduct, (Sym::Prod<Sym::Add<Sym::Var, Sym::Num>, Sym::Var, Sym::Pow<Sym::E, Sym::Var>>), "(x+5.6)*(x*e^x)", 5.6)
    META_TEST_INIT(EToXTower, (Sym::Pow<Sym::E, Sym::Pow<Sym::E, Sym::Pow<Sym::E, Sym::Pow<Sym::E, Sym::Pow<Sym::E, Sym::Pow<Sym::E, Sym::Var>>>>>>), "e^e^e^e^e^e^x")
    // solution, candidate, integral, vacancy, singleIntegralVacancy
    
    // From::Create::WithMap
    TEST(MetaOperatorsInitTest, FromCreateWithMap) { // NOLINT
        auto expression = Parser::parse_function("x+4+x^6+e^(2*x)+9+cos(sin(x))+2*u");
        auto expected_expression = Parser::parse_function(
            "arcsin(2*u)*arcsin(cos(sin(x)))*arcsin(9)*arcsin(e^(2*x))*arcsin(x^6)*arcsin(4)*arcsin(x)");

        size_t const count = expression.data()->as<Sym::Addition>().tree_size();
        std::vector<Sym::Symbol> destination(Sym::EXPRESSION_MAX_SYMBOL_COUNT);

        Sym::From<Sym::Addition>::Create<Sym::Product>::WithMap<Sym::Arcsine>::init(
            *destination.data(), {{expression.data()->as<Sym::Addition>(), count}});

        destination.resize(destination.data()->size());

        EXPECT_TRUE(Sym::Symbol::compare_trees(destination.data(), expected_expression.data()));
    }

    // Match
    META_TEST_MATCH(Variable, Sym::Var, "x")
    META_TEST_MATCH(Pi, Sym::Pi, "pi")
    META_TEST_MATCH(E, Sym::E, "e")
    META_TEST_MATCH(Integer, Sym::Integer<69>, "69")
    META_TEST_MATCH(NumericConstant, Sym::Num, "123.456")
    // Const
    META_TEST_MATCH(ConstantExpression, Sym::Const, "sin(e)+pi+c^456+cos(tan(1))*(-ln(2))")
    META_TEST_NOT_MATCH(NotConstantExpression, Sym::Const, "e^pi+sin(x)")
    // Simple OneArgOperators
    META_TEST_MATCH(Sine, Sym::Sin<Sym::E>, "sin(e)")
    META_TEST_MATCH(Cosine, Sym::Cos<Sym::Var>, "cos(x)")
    META_TEST_MATCH(Tangent, Sym::Tan<Sym::Pi>, "tan(pi)")
    META_TEST_MATCH(Cotangent, Sym::Cot<Sym::Sin<Sym::Var>>, "cot(sin(x))")
    META_TEST_MATCH(Arcsine, Sym::Arcsin<Sym::E>, "arcsin(e)")
    META_TEST_MATCH(Arccosine, Sym::Arccos<Sym::E>, "arccos(e)")
    META_TEST_MATCH(Arctangent, Sym::Arctan<Sym::E>, "arctan(e)")
    META_TEST_MATCH(Arccotangent, Sym::Arccot<Sym::E>, "arccot(e)")
    META_TEST_MATCH(Logarithm, Sym::Ln<Sym::Var>, "ln(x)")
    // Simple TwoArgOperators
    META_TEST_MATCH(Sum, (Sym::Add<Sym::Cos<Sym::E>, Sym::Pi>), "cos(e)+pi")
    META_TEST_MATCH(Product, (Sym::Mul<Sym::Cos<Sym::Var>, Sym::Pi>), "cos(x)*pi")
    META_TEST_MATCH(Power, (Sym::Pow<Sym::Cos<Sym::E>, Sym::Pi>), "cos(e)^pi")
    // AnyOf, AllOf, Not
    META_TEST_MATCH(AnyOfFirstCorrect, (Sym::AnyOf<Sym::Cos<Sym::Var>, Sym::E, Sym::Integer<3>>), "cos(x)")
    META_TEST_MATCH(AnyOfSecondCorrect, (Sym::AnyOf<Sym::Cos<Sym::Var>, Sym::E, Sym::Integer<3>>), "e")
    META_TEST_MATCH(AnyOfLastCorrect, (Sym::AnyOf<Sym::Cos<Sym::Var>, Sym::E, Sym::Integer<3>>), "3")
    META_TEST_NOT_MATCH(AnyOfNoneCorrect, (Sym::AnyOf<Sym::Cos<Sym::Var>, Sym::E, Sym::Integer<3>>), "sin(e)")

    META_TEST_NOT_MATCH(NotMatchAllOf, (Sym::AllOf<Sym::Cos<Sym::Var>, Sym::E, Sym::Integer<3>>), "e")
    META_TEST_MATCH(SingleAllOf, (Sym::AllOf<Sym::Cos<Sym::Var>>), "cos(x)")

    META_TEST_MATCH(NotMatchesWrongExpression, (Sym::Not<Sym::AllOf<Sym::Cos<Sym::Var>, Sym::Var>>), "cos(x)")
    META_TEST_NOT_MATCH(NotWithTrueCondition, (Sym::Not<Sym::Arcsin<Sym::E>>), "arcsin(e)")
    // Same, PatternPair
    META_TEST_MATCH(SimpleSame, (Sym::Mul<Sym::Same, Sym::Same>), "(e^x*345+1)*(e^x*345+1)")
    META_TEST_NOT_MATCH(NotSame, (Sym::Mul<Sym::Same, Sym::Same>), "(e^x*345+1)*(e^x*345)")
    META_TEST_MATCH(AdvancedSame, (Sym::Add<Sym::Ln<Sym::Mul<Sym::Same, Sym::Num>>, Sym::Sin<Sym::Add<Sym::E, Sym::Same>>>), "ln((x+sin(x)+4^x)*5.7)+sin(e+(x+sin(x)+4^x))")
    META_TEST_MATCH(FourSameSymbols, (Sym::Mul<Sym::Add<Sym::Same, Sym::Same>, Sym::Add<Sym::Same, Sym::Mul<Sym::Integer<4>, Sym::Same>>>), "(e^c^x+e^c^x)*(e^c^x+4*e^c^x)")

    META_TEST_MATCH_PAIR(PairWithIndependentPatterns, Sym::Ln<Sym::Var>, Sym::Arccot<Sym::E>, "ln(x)", "arccot(e)")
    // Advanced expressions
    META_TEST_MATCH(LongSum, (Sym::Sum<Sym::Var, Sym::Cos<Sym::Ln<Sym::Const>>, Sym::E, Sym::Integer<1>>), "x+(cos(ln(2+3+c))+(e+1))")
    META_TEST_MATCH(LongProduct, (Sym::Prod<Sym::Add<Sym::Var, Sym::Num>, Sym::Var, Sym::Pow<Sym::E, Sym::Var>>), "(x+5.6)*(x*e^x)")
    META_TEST_MATCH(EToXTower, (Sym::Pow<Sym::E, Sym::Pow<Sym::E, Sym::Pow<Sym::E, Sym::Pow<Sym::E, Sym::Pow<Sym::E, Sym::Pow<Sym::E, Sym::Var>>>>>>), "e^e^e^e^e^e^x")
        
    // solution, candidate, integral, vacancy, singleIntegralVacancy
    
}

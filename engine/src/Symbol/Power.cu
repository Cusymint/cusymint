#include "hip/hip_runtime.h"
#include "Power.cuh"

#include "Symbol.cuh"
#include <fmt/core.h>

namespace Sym {
    DEFINE_TWO_ARGUMENT_OP_FUNCTIONS(Power)
    DEFINE_SIMPLE_TWO_ARGUMENT_OP_COMPARE(Power)
    DEFINE_TWO_ARGUMENT_OP_COMPRESS_REVERSE_TO(Power)

    DEFINE_SIMPLIFY_IN_PLACE(Power) {
        arg1().simplify_in_place(help_space);
        arg2().simplify_in_place(help_space);

        if (arg2().is(Type::NumericConstant) && arg2().numeric_constant.value == 0.0) {
            Symbol::from(this)->numeric_constant = NumericConstant::with_value(1.0);
            return;
        }

        if (arg1().is(Type::NumericConstant) && arg2().is(Type::NumericConstant)) {
            double value1 = arg1().numeric_constant.value;
            double value2 = arg2().numeric_constant.value;
            Symbol::from(this)->numeric_constant = NumericConstant::with_value(pow(value1, value2));
            return;
        }

        // (a^b)^c -> a^(b*c)
        if (arg1().is(Type::Power)) {
            Symbol::from(this)->copy_to(help_space);
            Power* const this_copy = &help_space->power;

            *this = Power::create();
            this_copy->arg1().power.arg1().copy_to(&arg1());
            seal_arg1();

            Product* const product = &arg2() << Product::create();
            this_copy->arg1().power.arg2().copy_to(&product->arg1());
            product->seal_arg1();
            this_copy->arg2().copy_to(&product->arg2());
            product->seal();

            seal();

            return;
        }
    }

    DEFINE_IS_FUNCTION_OF(Power) {
        for (size_t i = 0; i < expression_count; ++i) {
            if (!expressions[i]->is(Type::Power)) {
                continue;
            }

            const auto& power_expression = expressions[i]->as<Power>();

            // TODO: In the future, this should look for correspondences in the product tree of
            // arg1(). For example, if `this` is `e^(pi*x*2*sin(x))`, then this function should
            // return true when `power_expression` is `e^(sin(x)*x)`.
            if (arg1() == power_expression.arg1() && arg2() == power_expression.arg2()) {
                return true;
            }
        }

        return arg1().is_function_of(expressions, expression_count) &&
               arg2().is_function_of(expressions, expression_count);
    }

    std::string Power::to_string() const {
        return fmt::format("({}^{})", arg1().to_string(), arg2().to_string());
    }

    std::string Power::to_tex() const {
        if (arg1().is(Type::Addition) || arg1().is(Type::Product) || arg1().is(Type::Negation) ||
            arg1().is(Type::Reciprocal) || arg1().is(Type::Power)) {
            return fmt::format(R"(\left({}\right)^{{ {} }})", arg1().to_tex(), arg2().to_tex());
        }
        return fmt::format("{}^{{ {} }}", arg1().to_tex(), arg2().to_tex());
    }

    std::vector<Symbol> operator^(const std::vector<Symbol>& lhs, const std::vector<Symbol>& rhs) {
        std::vector<Symbol> res(lhs.size() + rhs.size() + 1);
        Power::create(lhs.data(), rhs.data(), res.data());
        return res;
    }
}

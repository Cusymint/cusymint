#include "hip/hip_runtime.h"
#include "Trigonometric.cuh"

#include "Symbol.cuh"
#include <fmt/core.h>

namespace Sym {
    DEFINE_ONE_ARGUMENT_OP_FUNCTIONS(Sine)
    DEFINE_SIMPLE_ONE_ARGUMETN_OP_COMPARE(Sine)
    DEFINE_ONE_ARGUMENT_OP_COMPRESS_REVERSE_TO(Sine)
    DEFINE_SIMPLE_ONE_ARGUMENT_IS_FUNCTION_OF(Sine)

    DEFINE_SIMPLIFY_IN_PLACE(Sine) {
        arg().simplify_in_place(help_space);

        if (arg().is(Type::Arcsine)) {
            arg().as<Arcsine>().arg().copy_to(help_space);
            help_space->copy_to(symbol());
        }
    }

    DEFINE_ONE_ARGUMENT_OP_FUNCTIONS(Cosine)
    DEFINE_SIMPLE_ONE_ARGUMETN_OP_COMPARE(Cosine)
    DEFINE_ONE_ARGUMENT_OP_COMPRESS_REVERSE_TO(Cosine)
    DEFINE_SIMPLE_ONE_ARGUMENT_IS_FUNCTION_OF(Cosine)

    DEFINE_SIMPLIFY_IN_PLACE(Cosine) {
        arg().simplify_in_place(help_space);

        if (arg().is(Type::Arccosine)) {
            arg().as<Arccosine>().arg().copy_to(help_space);
            help_space->copy_to(symbol());
        }
    }

    DEFINE_ONE_ARGUMENT_OP_FUNCTIONS(Tangent)
    DEFINE_SIMPLE_ONE_ARGUMETN_OP_COMPARE(Tangent)
    DEFINE_ONE_ARGUMENT_OP_COMPRESS_REVERSE_TO(Tangent)
    DEFINE_SIMPLE_ONE_ARGUMENT_IS_FUNCTION_OF(Tangent)

    DEFINE_SIMPLIFY_IN_PLACE(Tangent) {
        arg().simplify_in_place(help_space);

        if (arg().is(Type::Arctangent)) {
            arg().as<Arctangent>().arg().copy_to(help_space);
            help_space->copy_to(symbol());
        }
    }

    DEFINE_ONE_ARGUMENT_OP_FUNCTIONS(Cotangent)
    DEFINE_SIMPLE_ONE_ARGUMETN_OP_COMPARE(Cotangent)
    DEFINE_ONE_ARGUMENT_OP_COMPRESS_REVERSE_TO(Cotangent)
    DEFINE_SIMPLE_ONE_ARGUMENT_IS_FUNCTION_OF(Cotangent)

    DEFINE_SIMPLIFY_IN_PLACE(Cotangent) {
        arg().simplify_in_place(help_space);

        if (arg().is(Type::Arccotangent)) {
            arg().as<Arccotangent>().arg().copy_to(help_space);
            help_space->copy_to(symbol());
        }
    }

    std::string Sine::to_string() const { return fmt::format("sin({})", arg().to_string()); }

    std::string Cosine::to_string() const { return fmt::format("cos({})", arg().to_string()); }

    std::string Tangent::to_string() const { return fmt::format("tan({})", arg().to_string()); }

    std::string Cotangent::to_string() const { return fmt::format("cot({})", arg().to_string()); }

    std::string Sine::to_tex() const {
        return fmt::format(R"(\sin\left({}\right))", arg().to_tex());
    }

    std::string Cosine::to_tex() const {
        return fmt::format(R"(\cos\left({}\right))", arg().to_tex());
    }

    std::string Tangent::to_tex() const {
        return fmt::format(R"(\tan\left({}\right))", arg().to_tex());
    }

    std::string Cotangent::to_tex() const {
        return fmt::format(R"(\cot\left({}\right))", arg().to_tex());
    }

    template <class T> std::vector<Symbol> make_trig_function(const std::vector<Symbol>& arg) {
        std::vector<Symbol> res(arg.size() + 1);
        T::create(arg.data(), res.data());
        return res;
    }

    std::vector<Symbol> sin(const std::vector<Symbol>& arg) {
        return make_trig_function<Sine>(arg);
    }

    std::vector<Symbol> cos(const std::vector<Symbol>& arg) {
        return make_trig_function<Cosine>(arg);
    }

    std::vector<Symbol> tan(const std::vector<Symbol>& arg) {
        return make_trig_function<Tangent>(arg);
    }

    std::vector<Symbol> cot(const std::vector<Symbol>& arg) {
        return make_trig_function<Cotangent>(arg);
    }
}

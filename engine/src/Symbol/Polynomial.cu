#include "hip/hip_runtime.h"
#include "Macros.cuh"
#include "Polynomial.cuh"
#include "Symbol.cuh"
#include "Utils/Cuda.cuh"
#include <fmt/core.h>

namespace Sym {
    DEFINE_COMPRESS_REVERSE_TO(Polynomial) {
        Symbol::copy_and_reverse_symbol_sequence(destination, this_symbol(), size);
        return size;
    }

    DEFINE_NO_OP_SIMPLIFY_IN_PLACE(Polynomial)
    DEFINE_INTO_DESTINATION_OPERATOR(Polynomial)

    __host__ __device__ bool are_coefficients_equal(const Polynomial& poly1,
                                                    const Polynomial& poly2) {
        // Assumption: polynomials have the same rank
        for (int i = 0; i <= poly1.rank; ++i) {
            if (poly1[i] != poly2[i]) {
                return false;
            }
        }
        return true;
    }

    DEFINE_COMPARE(Polynomial) {
        return BASE_COMPARE(Polynomial) && symbol->polynomial.rank == rank &&
               are_coefficients_equal(*this, symbol->polynomial);
    }

    __host__ __device__ Polynomial Polynomial::with_rank(int rank) {
        return {
            .type = Type::Polynomial,
            .size = 2 + sizeof(double) * (rank + 1) / sizeof(Symbol),
            .simplified = true,
            .rank = rank,
        };
    }

    __host__ __device__ void Polynomial::divide_polynomials(Polynomial& numerator,
                                                            Polynomial& denominator,
                                                            Polynomial& result) {
        for (int i = numerator.rank - denominator.rank; i >= 0; --i) {
            double& num_first = numerator[i + denominator.rank];
            double& res_current = result[i];
            res_current = num_first / denominator[denominator.rank];
            num_first = 0;
            for (int j = denominator.rank - 1; j >= 0; --j) {
                numerator[i + j] -= res_current * denominator[j];
            }
        }
        numerator.make_proper();
    }

    __host__ __device__ void Polynomial::make_proper() {
        int i = rank;
        while (i >= 0 && abs(coefficients()[i--]) < Util::eps) {
            --rank;
        }
        size = 2 + sizeof(double) * (rank + 1) / sizeof(Symbol);
    }

    std::string Polynomial::to_string() const { // TODO lepiej!
        std::string coefficients_str =
            fmt::format(rank == 0 ? "Poly[size={}]({}" : (rank == 1 ? "Poly[size={}]({}^x" : "Poly[size={}]({}x^({})"), size, coefficients()[rank], rank);
        for (int i = rank - 1; i > 1; --i) {
            if (coefficients()[i] != 0) {
                coefficients_str += fmt::format("{}{}*x^({})", coefficients()[i] < 0 ? "" : "+",
                                                coefficients()[i], i);
            }
        }
        if (rank > 1 && coefficients()[1] != 0) {
            coefficients_str +=
                fmt::format("{}{}*x", coefficients()[1] < 0 ? "" : "+", coefficients()[1]);
        }
        if (rank > 0 && coefficients()[0] != 0) {
            coefficients_str +=
                fmt::format("{}{}", coefficients()[0] < 0 ? "" : "+", coefficients()[0]);
        }
        return coefficients_str + ")";
    }

    std::string Polynomial::to_tex() const { // TODO popraw!
        std::string coefficients_str = fmt::format("{}x^{{ {} }}", coefficients()[rank], rank);
        for (int i = rank - 1; i >= 0; --i) {
            coefficients_str += fmt::format("{}{}x^{{ {} }}", coefficients()[i] < 0 ? "" : "+",
                                            coefficients()[i], i);
        }
        if (rank > 0 && coefficients()[1] != 0) {
            coefficients_str +=
                fmt::format("{}{}x", coefficients()[1] < 0 ? "" : "+", coefficients()[1]);
        }
        if (coefficients()[0] != 0) {
            coefficients_str +=
                fmt::format("{}{}", coefficients()[0] < 0 ? "" : "+", coefficients()[0]);
        }
        return coefficients_str;
    }

    __host__ __device__ double* Polynomial::coefficients() {
        return reinterpret_cast<double*>(&(Symbol::from(this)[1]));
    }

    __host__ __device__ const double* Polynomial::coefficients() const {
        return reinterpret_cast<const double*>(&(Symbol::from(this)[1]));
    }
}
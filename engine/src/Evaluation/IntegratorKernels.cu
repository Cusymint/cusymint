#include "hip/hip_runtime.h"
#include "IntegratorKernels.cuh"

#include "Heuristic/Heuristic.cuh"
#include "KnownIntegral/KnownIntegral.cuh"

namespace Sym {
    __device__ bool is_nonzero(const size_t index,
                               const Util::DeviceArray<uint32_t>& inclusive_scan) {
        return index == 0 && inclusive_scan[index] != 0 ||
               index != 0 && inclusive_scan[index - 1] != inclusive_scan[index];
    }
}

namespace Sym::Kernel {
    namespace {
        constexpr size_t TRANSFORM_GROUP_SIZE = 32;
    }

    /*
     * @brief Try to set `expressions[potential_solver_idx]` (SubexpressionCandidate)
     * as a solution to its SubexpressionVacancy
     *
     * @param expressions Expressions array with a candidate to solve and a missing
     * subexpression
     * @param potential_solver_idx Index of the potential solver
     *
     * @return `false` when haven't managed to set chosen candidate as a solution to
     * the subexpression or whetether there are still unsolved subexpressions in the parent.
     * `true` when managed to set chosen candidate as a solution and parent doesn't have any
     * unsolved subexpressions left.
     */
    __device__ bool try_set_solver_idx(Sym::ExpressionArray<>& expressions,
                                       const size_t potential_solver_idx) {
        const size_t& vacancy_expr_idx =
            expressions[potential_solver_idx].subexpression_candidate.vacancy_expression_idx;

        const size_t& vacancy_idx =
            expressions[potential_solver_idx].subexpression_candidate.vacancy_idx;

        Sym::SubexpressionVacancy& subexpr_vacancy =
            expressions[vacancy_expr_idx][vacancy_idx].subexpression_vacancy;

        const bool solver_lock_acquired = atomicCAS(&subexpr_vacancy.is_solved, 0, 1) == 0;

        if (!solver_lock_acquired) {
            return false;
        }

        subexpr_vacancy.solver_idx = potential_solver_idx;

        if (!expressions[vacancy_expr_idx].is(Sym::Type::SubexpressionCandidate)) {
            return true;
        }

        unsigned int subexpressions_left = atomicSub(
            &expressions[vacancy_expr_idx].subexpression_candidate.subexpressions_left, 1) - 1;

        return subexpressions_left == 0;
    }

    /*
     * @brief Sets `var` to `val` atomically
     *
     * @brief var Variable to set
     * @brief val Value assigned to `var`
     *
     * @return `false` if `var` was already equal to `val`, `true` otherwise
     */
    template <class T> __device__ bool try_set(T& var, const T& val) {
        const unsigned int previous_val = atomicExch(&var, val);
        return previous_val != val;
    }

    /*
     * @brief Gets target index from `scan` inclusive scan array at `index` index
     */
    __device__ uint32_t index_from_scan(const Util::DeviceArray<uint32_t>& scan,
                                        const size_t index) {
        if (index == 0) {
            return 0;
        }

        return scan[index - 1];
    }

    __global__ void simplify(const ExpressionArray<> expressions, ExpressionArray<> destination,
                             ExpressionArray<> help_spaces) {
        const size_t thread_count = Util::thread_count();
        const size_t thread_idx = Util::thread_idx();

        for (size_t expr_idx = thread_idx; expr_idx < expressions.size();
             expr_idx += thread_count) {
            expressions[expr_idx].copy_to(&destination[expr_idx]);
            destination[expr_idx].simplify(help_spaces.at(expr_idx));
        }
    }

    __global__ void
    check_for_known_integrals(const ExpressionArray<SubexpressionCandidate> integrals,
                              Util::DeviceArray<uint32_t> applicability) {
        const size_t thread_count = Util::thread_count();
        const size_t thread_idx = Util::thread_idx();

        const size_t check_step = thread_count / TRANSFORM_GROUP_SIZE;

        for (size_t check_idx = thread_idx / TRANSFORM_GROUP_SIZE; check_idx < KnownIntegral::COUNT;
             check_idx += check_step) {
            for (size_t int_idx = thread_idx % TRANSFORM_GROUP_SIZE; int_idx < integrals.size();
                 int_idx += TRANSFORM_GROUP_SIZE) {
                size_t appl_idx = MAX_EXPRESSION_COUNT * check_idx + int_idx;
                applicability[appl_idx] =
                    KnownIntegral::CHECKS[check_idx](integrals[int_idx].arg().as<Integral>());
            }
        }
    }

    __global__ void apply_known_integrals(const ExpressionArray<SubexpressionCandidate> integrals,
                                          ExpressionArray<> expressions,
                                          ExpressionArray<> help_spaces,
                                          const Util::DeviceArray<uint32_t> applicability) {
        const size_t thread_count = Util::thread_count();
        const size_t thread_idx = Util::thread_idx();

        const size_t trans_step = thread_count / TRANSFORM_GROUP_SIZE;

        for (size_t trans_idx = thread_idx / TRANSFORM_GROUP_SIZE; trans_idx < KnownIntegral::COUNT;
             trans_idx += trans_step) {
            for (size_t int_idx = thread_idx % TRANSFORM_GROUP_SIZE; int_idx < integrals.size();
                 int_idx += TRANSFORM_GROUP_SIZE) {
                const size_t appl_idx = MAX_EXPRESSION_COUNT * trans_idx + int_idx;

                if (!is_nonzero(appl_idx, applicability)) {
                    continue;
                }

                const size_t dest_idx =
                    expressions.size() + index_from_scan(applicability, appl_idx);

                auto* const subexpr_candidate = expressions.at(dest_idx)
                                                << SubexpressionCandidate::builder();
                subexpr_candidate->copy_metadata_from(integrals[int_idx]);
                KnownIntegral::APPLICATIONS[trans_idx](integrals[int_idx].arg().as<Integral>(),
                                                       subexpr_candidate->arg(),
                                                       help_spaces[dest_idx]);
                subexpr_candidate->seal();

                try_set_solver_idx(expressions, dest_idx);
            }
        }
    }

    __global__ void propagate_solved_subexpressions(ExpressionArray<> expressions) {
        const size_t thread_count = Util::thread_count();
        const size_t thread_idx = Util::thread_idx();

        // For each tree node there is a seperate starting thread.
        // If its node is solved it moves to it's parent.
        // It tries to fill the parent's vacancy with it's own solution.
        // If it succeeds and all of the parent's vacancies are solved, it moves to the parent.
        // This operation upwards is repeated upwards while all solutions to the current node
        // exists and the parent's vacancy is not solved and ends at the root.

        // Since `expr_idx = 0` is SubexpressionVacancy of the original integral, it is skipped
        for (size_t expr_idx = thread_idx + 1; expr_idx < expressions.size();
             expr_idx += thread_count) {
            size_t current_expr_idx = expr_idx;
            while (current_expr_idx != 0) {
                printf("%lu\n",current_expr_idx);
                if (expressions[current_expr_idx].subexpression_candidate.subexpressions_left !=
                    0) {
                    break;
                }

                if (!try_set_solver_idx(expressions, current_expr_idx)) {
                    break;
                }

                // We iterate tree upwards.
                // It may seem that there is a possibility of race condition
                // when we will reach the same node, as the thread which has started the loop.
                // However, since `try_set_solver_idx` is atomic, only one thread would be able
                // to set `solver_idx` on the next parent and continue its journey upwards.
                current_expr_idx =
                    expressions[current_expr_idx].subexpression_candidate.vacancy_expression_idx;
            }
        }
    }

    __global__ void find_redundand_expressions(const ExpressionArray<> expressions,
                                               Util::DeviceArray<uint32_t> removability) {
        const size_t thread_count = Util::thread_count();
        const size_t thread_idx = Util::thread_idx();

        // Look further and further in the dependency tree and check whether we are not trying
        // to solve something that has been solved already
        for (size_t expr_idx = thread_idx; expr_idx < expressions.size();
             expr_idx += thread_count) {
            removability[expr_idx] = 1;
            size_t current_expr_idx = expr_idx;

            while (current_expr_idx != 0) {
                const size_t& parent_idx =
                    expressions[current_expr_idx].subexpression_candidate.vacancy_expression_idx;
                const size_t& parent_vacancy_idx =
                    expressions[current_expr_idx].subexpression_candidate.vacancy_idx;
                const SubexpressionVacancy& parent_vacancy =
                    expressions[parent_idx][parent_vacancy_idx].subexpression_vacancy;

                if (parent_vacancy.is_solved == 1 &&
                    parent_vacancy.solver_idx != current_expr_idx) {
                    removability[expr_idx] = 0;
                    break;
                }

                current_expr_idx = parent_idx;
            }
        }
    }

    __global__ void
    find_redundand_integrals(const ExpressionArray<> integrals, const ExpressionArray<> expressions,
                             const Util::DeviceArray<uint32_t> expressions_removability,
                             Util::DeviceArray<uint32_t> integrals_removability) {
        const size_t thread_count = Util::thread_count();
        const size_t thread_idx = Util::thread_idx();

        for (size_t int_idx = thread_idx; int_idx < integrals.size(); int_idx += thread_count) {
            const size_t& vacancy_expr_idx =
                integrals[int_idx].subexpression_candidate.vacancy_expression_idx;
            const size_t& vacancy_idx = integrals[int_idx].subexpression_candidate.vacancy_idx;

            const bool parent_expr_failed = expressions_removability[vacancy_expr_idx] == 0;
            const bool parent_vacancy_solved =
                expressions[vacancy_expr_idx][vacancy_idx].subexpression_vacancy.is_solved == 1;

            integrals_removability[int_idx] = parent_expr_failed || parent_vacancy_solved ? 0 : 1;
        }
    }

    __global__ void remove_integrals(const ExpressionArray<SubexpressionCandidate> integrals,
                                     const Util::DeviceArray<uint32_t> integrals_removability,
                                     const Util::DeviceArray<uint32_t> expressions_removability,
                                     ExpressionArray<> destinations) {
        const size_t thread_count = Util::thread_count();
        const size_t thread_idx = Util::thread_idx();

        for (size_t int_idx = thread_idx; int_idx < integrals.size(); int_idx += thread_count) {
            if (!is_nonzero(int_idx, integrals_removability)) {
                continue;
            }

            Symbol& destination = destinations[integrals_removability[int_idx] - 1];
            integrals[int_idx].symbol()->copy_to(&destination);

            size_t& vacancy_expr_idx =
                destination.as<SubexpressionCandidate>().vacancy_expression_idx;
            vacancy_expr_idx = expressions_removability[vacancy_expr_idx] - 1;
        }
    }

    __global__ void
    check_heuristics_applicability(const ExpressionArray<SubexpressionCandidate> integrals,
                                   ExpressionArray<> expressions,
                                   Util::DeviceArray<uint32_t> new_integrals_flags,
                                   Util::DeviceArray<uint32_t> new_expressions_flags) {
        const size_t thread_count = Util::thread_count();
        const size_t thread_idx = Util::thread_idx();

        const size_t check_step = thread_count / TRANSFORM_GROUP_SIZE;

        for (size_t check_idx = thread_idx / TRANSFORM_GROUP_SIZE; check_idx < Heuristic::COUNT;
             check_idx += check_step) {
            for (size_t int_idx = thread_idx % TRANSFORM_GROUP_SIZE; int_idx < integrals.size();
                 int_idx += TRANSFORM_GROUP_SIZE) {
                size_t appl_idx = MAX_EXPRESSION_COUNT * check_idx + int_idx;
                Heuristic::CheckResult result =
                    Heuristic::CHECKS[check_idx](integrals[int_idx].arg().as<Integral>());
                new_integrals_flags[appl_idx] = result.new_integrals;
                new_expressions_flags[appl_idx] = result.new_expressions;

                const size_t& vacancy_expr_idx = integrals[int_idx].vacancy_expression_idx;
                const size_t& vacancy_idx = integrals[int_idx].vacancy_idx;
                SubexpressionVacancy& parent_vacancy =
                    expressions[vacancy_expr_idx][vacancy_idx].subexpression_vacancy;

                if (result.new_expressions == 0) {
                    // Assume new integrals are direct children of the vacancy
                    atomicAdd(&parent_vacancy.candidate_integral_count, result.new_integrals);
                }
                else {
                    // Assume new integrals are going to be children of new expressions, which
                    // are going to be children of the vacancy
                    atomicAdd(&parent_vacancy.candidate_expression_count, result.new_expressions);
                }
            }
        }
    }

    __global__ void apply_heuristics(const ExpressionArray<SubexpressionCandidate> integrals,
                                     ExpressionArray<> integrals_destinations,
                                     ExpressionArray<> expressions_destinations,
                                     ExpressionArray<> help_spaces,
                                     const Util::DeviceArray<uint32_t> new_integrals_indices,
                                     const Util::DeviceArray<uint32_t> new_expressions_indices) {
        const size_t thread_count = Util::thread_count();
        const size_t thread_idx = Util::thread_idx();

        const size_t trans_step = thread_count / TRANSFORM_GROUP_SIZE;

        for (size_t trans_idx = thread_idx / TRANSFORM_GROUP_SIZE; trans_idx < Heuristic::COUNT;
             trans_idx += trans_step) {
            for (size_t int_idx = thread_idx % TRANSFORM_GROUP_SIZE; int_idx < integrals.size();
                 int_idx += TRANSFORM_GROUP_SIZE) {
                const size_t appl_idx = MAX_EXPRESSION_COUNT * trans_idx + int_idx;
                if (!is_nonzero(appl_idx, new_integrals_indices)) {
                    continue;
                }

                const size_t int_dst_idx = index_from_scan(new_integrals_indices, appl_idx);

                if (new_expressions_indices[appl_idx] != 0) {
                    const size_t expr_dst_idx = expressions_destinations.size() +
                                                index_from_scan(new_expressions_indices, appl_idx);
                    Heuristic::APPLICATIONS[trans_idx](
                        integrals[int_idx], integrals_destinations.iterator(int_dst_idx),
                        expressions_destinations.iterator(expr_dst_idx), help_spaces[int_dst_idx]);
                }
                else {
                    Heuristic::APPLICATIONS[trans_idx](
                        integrals[int_idx], integrals_destinations.iterator(int_dst_idx),
                        ExpressionArray<>::Iterator::null(), help_spaces[int_dst_idx]);
                }
            }
        }
    }

    __global__ void propagate_failures_upwards(ExpressionArray<> expressions,
                                               Util::DeviceArray<uint32_t> failures) {
        const size_t thread_count = Util::thread_count();
        const size_t thread_idx = Util::thread_idx();

        for (size_t expr_idx = thread_idx; expr_idx < expressions.size();
             expr_idx += thread_count) {
            SubexpressionCandidate& self_candidate = expressions[expr_idx].subexpression_candidate;

            // Some other thread was here already, as `failures` starts with 1 everywhere
            if (failures[expr_idx] == 0) {
                continue;
            }

            bool is_failed = false;

            // expressions[current_expr_idx][0] is subexpression_candidate, so it could be
            // skipped, but if `expr_idx == 0` it is the only SubexpressionVacancy
            for (size_t sym_idx = 0; sym_idx < expressions[expr_idx].size(); ++sym_idx) {
                if (!expressions[expr_idx][sym_idx].is(Type::SubexpressionVacancy)) {
                    continue;
                }

                SubexpressionVacancy& vacancy =
                    expressions[expr_idx][sym_idx].subexpression_vacancy;

                if (vacancy.candidate_integral_count == 0 &&
                    vacancy.candidate_expression_count == 0 && vacancy.is_solved == 0) {
                    is_failed = true;
                    break;
                }
            }

            if (!is_failed || !try_set(failures[expr_idx], 0U)) {
                continue;
            }

            size_t current_expr_idx = expr_idx;
            while (current_expr_idx != 0) {
                const size_t& parent_idx =
                    expressions[current_expr_idx].subexpression_candidate.vacancy_expression_idx;
                const size_t& vacancy_idx =
                    expressions[current_expr_idx].subexpression_candidate.vacancy_idx;
                SubexpressionVacancy& parent_vacancy =
                    expressions[parent_idx][vacancy_idx].subexpression_vacancy;

                if (parent_vacancy.candidate_integral_count != 0 || parent_vacancy.is_solved == 1) {
                    break;
                }

                const size_t parent_vacancy_candidates_left =
                    atomicSub(&parent_vacancy.candidate_expression_count, 1) - 1;

                // Go upwards if parent is failed
                if (parent_vacancy_candidates_left != 0 || !try_set(failures[parent_idx], 0U)) {
                    break;
                }

                current_expr_idx = parent_idx;
            }
        }
    }

    __global__ void propagate_failures_downwards(ExpressionArray<> expressions,
                                                 Util::DeviceArray<uint32_t> failures) {
        const size_t thread_count = Util::thread_count();
        const size_t thread_idx = Util::thread_idx();

        // Top expression has no parents, so we skip it
        for (size_t expr_idx = thread_idx + 1; expr_idx < expressions.size();
             expr_idx += thread_count) {
            size_t current_expr_idx = expr_idx;

            while (current_expr_idx != 0) {
                const size_t& parent_idx =
                    expressions[current_expr_idx].subexpression_candidate.vacancy_expression_idx;

                if (failures[parent_idx] == 0) {
                    failures[expr_idx] = 0;
                    break;
                }

                current_expr_idx = parent_idx;
            }
        }
    }

    __global__ void
    find_redundand_integrals(const ExpressionArray<> integrals,
                             const Util::DeviceArray<uint32_t> expressions_removability,
                             Util::DeviceArray<uint32_t> integrals_removability) {
        const size_t thread_count = Util::thread_count();
        const size_t thread_idx = Util::thread_idx();

        for (size_t int_idx = thread_idx; int_idx < integrals.size(); int_idx += thread_count) {
            const size_t& parent_idx =
                integrals[int_idx].subexpression_candidate.vacancy_expression_idx;

            integrals_removability[int_idx] = expressions_removability[parent_idx];
        }
    }

}

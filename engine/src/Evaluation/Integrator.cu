#include "Integrator.cuh"

#include <thrust/execution_policy.h>
#include <thrust/scan.h>

#include "IntegratorKernels.cuh"
#include "StaticFunctions.cuh"

#include "Utils/CompileConstants.cuh"
#include "Utils/Meta.cuh"

namespace Sym {
    std::vector<Sym::Symbol>
    Integrator::replace_nth_with_tree(std::vector<Sym::Symbol> expression, const size_t n,
                                      const std::vector<Sym::Symbol>& tree) {
        if constexpr (Consts::DEBUG) {
            if (!tree[0].is(Type::SubexpressionCandidate)) {
                Util::crash("Invalid first symbol of tree: %s, should be SubexpressionCandidate",
                            type_name(tree[0].type()));
            }
        }

        std::vector<Sym::Symbol> tree_content;

        if (tree[1].is(Sym::Type::Solution)) {
            tree_content = tree[1].as<Sym::Solution>().substitute_substitutions();
        }
        else {
            tree_content.resize(tree.size() - 1);
            std::copy(tree.begin() + 1, tree.end(), tree_content.begin());
        }

        expression[n].init_from(Sym::ExpanderPlaceholder::with_size(tree_content.size()));

        std::vector<Sym::Symbol> new_tree(expression.size() + tree_content.size() - 1);
        expression.data()->compress_to(*new_tree.data());

        std::copy(tree_content.begin(), tree_content.end(),
                  new_tree.begin() + static_cast<int64_t>(n));

        return new_tree;
    }

    std::vector<Sym::Symbol>
    Integrator::collapse_nth(const std::vector<std::vector<Sym::Symbol>>& tree, const size_t n) {
        std::vector<Sym::Symbol> current_collapse = tree[n];

        for (size_t i = 0; i < current_collapse.size(); ++i) {
            if (!current_collapse[i].is(Sym::Type::SubexpressionVacancy)) {
                continue;
            }

            const auto subtree =
                collapse_nth(tree, current_collapse[i].as<Sym::SubexpressionVacancy>().solver_idx);

            auto new_collapse = replace_nth_with_tree(current_collapse, i, subtree);
            i += new_collapse.size() - current_collapse.size();
            current_collapse = new_collapse;
        }

        return current_collapse;
    }

    std::vector<Sym::Symbol>
    Integrator::collapse(const std::vector<std::vector<Sym::Symbol>>& tree) {
        auto collapsed = collapse_nth(tree, 0);
        std::vector<Sym::Symbol> reversed(collapsed.size());
        const size_t new_size = collapsed.data()->compress_reverse_to(reversed.data());
        Sym::Symbol::copy_and_reverse_symbol_sequence(collapsed.data(), reversed.data(), new_size);

        std::vector<Sym::Symbol> help_space(EXPRESSION_MAX_SYMBOL_COUNT);
        collapsed.data()->simplify(help_space.data());
        collapsed.resize(collapsed.data()->size());

        return collapsed;
    }

    Integrator::Integrator() :
        MAX_CHECK_COUNT(KnownIntegral::COUNT > Heuristic::COUNT ? KnownIntegral::COUNT
                                                                : Heuristic::COUNT),
        SCAN_ARRAY_SIZE(MAX_CHECK_COUNT * MAX_EXPRESSION_COUNT),
        expressions(MAX_EXPRESSION_COUNT, EXPRESSION_MAX_SYMBOL_COUNT),
        expressions_swap(MAX_EXPRESSION_COUNT, EXPRESSION_MAX_SYMBOL_COUNT),
        integrals(MAX_EXPRESSION_COUNT, EXPRESSION_MAX_SYMBOL_COUNT),
        integrals_swap(MAX_EXPRESSION_COUNT, EXPRESSION_MAX_SYMBOL_COUNT),
        help_space(MAX_EXPRESSION_COUNT, EXPRESSION_MAX_SYMBOL_COUNT, integrals.size()),
        scan_array_1(SCAN_ARRAY_SIZE, true),
        scan_array_2(SCAN_ARRAY_SIZE, true) {}

    void Integrator::simplify_integrals() {
        integrals_swap.resize(integrals.size());
        Kernel::simplify<<<BLOCK_COUNT, BLOCK_SIZE>>>(integrals, integrals_swap, help_space);
        hipDeviceSynchronize();
        std::swap(integrals, integrals_swap);
    }

    void Integrator::check_for_known_integrals() {
        Kernel::check_for_known_integrals<<<BLOCK_COUNT, BLOCK_SIZE>>>(integrals, scan_array_1);
        hipDeviceSynchronize();

        thrust::inclusive_scan(thrust::device, scan_array_1.begin(), scan_array_1.end(),
                               scan_array_1.data());
        hipDeviceSynchronize();
    }

    void Integrator::apply_known_integrals() {
        Kernel::apply_known_integrals<<<BLOCK_COUNT, BLOCK_SIZE>>>(integrals, expressions,
                                                                   help_space, scan_array_1);
        hipDeviceSynchronize();
        expressions.increment_size_from_device(scan_array_1.last());

        Kernel::propagate_solved_subexpressions<<<BLOCK_COUNT, BLOCK_SIZE>>>(expressions);
        hipDeviceSynchronize();
    }

    bool Integrator::is_original_expression_solved() {
        std::vector<Symbol> first_expression = expressions.to_vector(0);
        return first_expression.data()->as<SubexpressionVacancy>().is_solved == 1;
    }

    void Integrator::remove_unnecessary_candidates() {
        scan_array_1.zero_mem();
        Kernel::find_redundand_expressions<<<BLOCK_COUNT, BLOCK_SIZE>>>(expressions, scan_array_1);
        hipDeviceSynchronize();

        Kernel::find_redundand_integrals<<<BLOCK_COUNT, BLOCK_SIZE>>>(integrals, expressions,
                                                                      scan_array_1, scan_array_2);
        hipDeviceSynchronize();

        thrust::inclusive_scan(thrust::device, scan_array_1.begin(), scan_array_1.end(),
                               scan_array_1.data());
        thrust::inclusive_scan(thrust::device, scan_array_2.begin(), scan_array_2.end(),
                               scan_array_2.data());
        hipDeviceSynchronize();

        Kernel::remove_expressions<true>
            <<<BLOCK_COUNT, BLOCK_SIZE>>>(expressions, scan_array_1, expressions_swap);
        Kernel::remove_integrals<<<BLOCK_COUNT, BLOCK_SIZE>>>(integrals, scan_array_2, scan_array_1,
                                                              integrals_swap);
        hipDeviceSynchronize();

        std::swap(expressions, expressions_swap);
        std::swap(integrals, integrals_swap);
        expressions.resize_from_device(scan_array_1.last());
        integrals.resize_from_device(scan_array_2.last());
    }

    void Integrator::check_heuristics_applicability() {
        scan_array_1.zero_mem();
        scan_array_2.zero_mem();
        Kernel::check_heuristics_applicability<<<BLOCK_COUNT, BLOCK_SIZE>>>(
            integrals, expressions, scan_array_1, scan_array_2);
        hipDeviceSynchronize();

        thrust::inclusive_scan(thrust::device, scan_array_1.begin(), scan_array_1.end(),
                               scan_array_1.data());
        thrust::inclusive_scan(thrust::device, scan_array_2.begin(), scan_array_2.end(),
                               scan_array_2.data());
        hipDeviceSynchronize();
    }

    void Integrator::apply_heuristics() {
        Kernel::apply_heuristics<<<BLOCK_COUNT, BLOCK_SIZE>>>(
            integrals, integrals_swap, expressions, help_space, scan_array_1, scan_array_2);
        hipDeviceSynchronize();

        std::swap(integrals, integrals_swap);
        integrals.resize_from_device(scan_array_1.last());
        expressions.increment_size_from_device(scan_array_2.last());

        scan_array_1.set_mem(1);
        Kernel::propagate_failures_upwards<<<BLOCK_COUNT, BLOCK_SIZE>>>(expressions, scan_array_1);
        hipDeviceSynchronize();
    }

    bool Integrator::has_original_expression_failed() { return scan_array_1.to_cpu(0) == 0; }

    void Integrator::remove_failed_candidates() {
        Kernel::propagate_failures_downwards<<<BLOCK_COUNT, BLOCK_SIZE>>>(expressions,
                                                                          scan_array_1);
        hipDeviceSynchronize();

        scan_array_2.zero_mem();
        Kernel::find_redundand_integrals<<<BLOCK_COUNT, BLOCK_SIZE>>>(integrals, scan_array_1,
                                                                      scan_array_2);
        hipDeviceSynchronize();

        thrust::inclusive_scan(thrust::device, scan_array_1.begin(), scan_array_1.end(),
                               scan_array_1.data());
        thrust::inclusive_scan(thrust::device, scan_array_2.begin(), scan_array_2.end(),
                               scan_array_2.data());
        hipDeviceSynchronize();

        Kernel::remove_expressions<false>
            <<<BLOCK_COUNT, BLOCK_SIZE>>>(expressions, scan_array_1, expressions_swap);
        Kernel::remove_integrals<<<BLOCK_COUNT, BLOCK_SIZE>>>(integrals, scan_array_2, scan_array_1,
                                                              integrals_swap);
        hipDeviceSynchronize();

        std::swap(expressions, expressions_swap);
        std::swap(integrals, integrals_swap);

        // How many expressions are left, cannot take scan_array_1.last() because space
        // after last place in scan_array_1 that corresponds to an expression is occupied
        // by ones
        expressions.resize(scan_array_1.to_cpu(expressions_swap.size() - 1));
        integrals.resize_from_device(scan_array_2.last());
        hipDeviceSynchronize();

        scan_array_1.zero_mem();
        scan_array_2.zero_mem();
    }

    std::optional<std::vector<Symbol>>
    Integrator::solve_integral(const std::vector<Symbol>& integral) {
        expressions.load_from_vector({single_integral_vacancy()});
        integrals.load_from_vector({first_expression_candidate(integral)});

        for (size_t i = 0;; ++i) {
            simplify_integrals();

            check_for_known_integrals();
            apply_known_integrals();

            if (is_original_expression_solved()) {
                return collapse(expressions.to_vector());
            }

            remove_unnecessary_candidates();

            check_heuristics_applicability();
            apply_heuristics();

            if (has_original_expression_failed()) {
                return std::nullopt;
            }

            remove_failed_candidates();
        }

        return std::nullopt;
    }
}

#include "hip/hip_runtime.h"
#include "Integrate.cuh"

#include <thrust/execution_policy.h>
#include <thrust/scan.h>

#include "Heuristic/Heuristic.cuh"
#include "KnownIntegral/KnownIntegral.cuh"
#include "StaticFunctions.cuh"

#include "Utils/CompileConstants.cuh"
#include "Utils/Cuda.cuh"
#include "Utils/Meta.cuh"

namespace Sym {
    namespace {
        constexpr size_t TRANSFORM_GROUP_SIZE = 32;
        constexpr size_t MAX_EXPRESSION_COUNT = 128;

        /*
         * @brief Try to set `expressions[potential_solver_idx]` (SubexpressionCandidate)
         * as a solution to its SubexpressionVacancy
         *
         * @param expressions Expressions array with a candidate to solve and a missing subexpression
         * @param potential_solver_idx Index of the potential solver
         *
         * @return `false` when haven't managed to set chosen candidate as a solution to
         * the subexpression or whetether there are still unsolved subexpressions in the parent.
         * `true` when managed to set chosen candidate as a solution and parent doesn't have any
         * unsolved subexpressions left. 
         */
        __device__ bool try_set_solver_idx(Sym::ExpressionArray<>& expressions,
                                           const size_t potential_solver_idx) {
            const size_t& vacancy_expr_idx =
                expressions[potential_solver_idx].subexpression_candidate.vacancy_expression_idx;

            const size_t& vacancy_idx =
                expressions[potential_solver_idx].subexpression_candidate.vacancy_idx;

            Sym::SubexpressionVacancy& subexpr_vacancy =
                expressions[vacancy_expr_idx][vacancy_idx].subexpression_vacancy;

            const bool solver_lock_acquired = atomicCAS(&subexpr_vacancy.is_solved, 0, 1) == 0;

            if (!solver_lock_acquired) {
                return false;
            }

            subexpr_vacancy.solver_idx = potential_solver_idx;

            if (!expressions[vacancy_expr_idx].is(Sym::Type::SubexpressionCandidate)) {
                return true;
            }

            unsigned int subexpressions_left = atomicSub(
                &expressions[vacancy_expr_idx].subexpression_candidate.subexpressions_left, 1);

            return subexpressions_left == 0;
        }

        /*
         * @brief Sets `var` to `val` atomically
         *
         * @brief var Variable to set
         * @brief val Value assigned to `var`
         *
         * @return `false` if `var` was already equal to `val`, `true` otherwise
         */
        template <class T> __device__ bool try_set(T& var, const T& val) {
            const unsigned int previous_val = atomicExch(&var, val);
            return previous_val != val;
        }

        /*
         * @brief Gets target index from `scan` inclusive scan array at `index` index
         */
        __device__ uint32_t index_from_scan(const Util::DeviceArray<uint32_t>& scan,
                                            const size_t index) {
            if (index == 0) {
                return 0;
            }

            return scan[index - 1];
        }

        /*
         * @brief Checks if inclusive_scan[index] is signaling a zero-sized element
         *
         * @param index Index to check
         * @param inclusive_scan Array of element sizes on which inclusive_scan has been run
         *
         * @return `false` if element is zero-sized, `true` otherwise
         */
        __device__ bool is_nonzero(const size_t index,
                                   const Util::DeviceArray<uint32_t>& inclusive_scan) {
            return index == 0 && inclusive_scan[index] != 0 ||
                   index != 0 && inclusive_scan[index - 1] != inclusive_scan[index];
        }

        /*
         * @brief Simplifies `expressions`. Result overrides `expressions` data.
         *
         * @param expressions Expressions to simplify
         * @param help_spaces Help space required for some simplifications
         */
        __global__ void simplify(ExpressionArray<> expressions, ExpressionArray<> help_spaces) {
            const size_t thread_count = Util::thread_count();
            const size_t thread_idx = Util::thread_idx();

            for (size_t expr_idx = thread_idx; expr_idx < expressions.size();
                 expr_idx += thread_count) {
                expressions[expr_idx].simplify(help_spaces.at(expr_idx));
            }
        }

        /*
         * @brief Checks whether `integrals` have known solutions
         *
         * @param integrals Integrals to be checked
         * @param applicability Solution to checking all `integrals` against known integrals.
         * `applicability[MAX_EXPRESSION_COUNT * form_idx + int_idx]` stores information whether
         * `KnownIntegral::APPLICATIONS[form_idx]` can be applied to `integral[int_idx]`, where
         * `MAX_EXPRESSION_COUNT` is the maximum size of the `integrals` array. 
         */
        __global__ void
        check_for_known_integrals(const ExpressionArray<SubexpressionCandidate> integrals,
                                  Util::DeviceArray<uint32_t> applicability) {
            const size_t thread_count = Util::thread_count();
            const size_t thread_idx = Util::thread_idx();

            const size_t check_step = thread_count / TRANSFORM_GROUP_SIZE;

            for (size_t check_idx = thread_idx / TRANSFORM_GROUP_SIZE;
                 check_idx < KnownIntegral::COUNT; check_idx += check_step) {
                for (size_t int_idx = thread_idx % TRANSFORM_GROUP_SIZE; int_idx < integrals.size();
                     int_idx += TRANSFORM_GROUP_SIZE) {
                    size_t appl_idx = MAX_EXPRESSION_COUNT * check_idx + int_idx;
                    applicability[appl_idx] =
                        KnownIntegral::CHECKS[check_idx](integrals[int_idx].arg().as<Integral>());
                }
            }
        }

        /*
         * @brief Solves integrals in place using the `applicability` information from `check_for_known_integrals`
         *
         * @param integrals Integrals with potentially known solutions
         * @param expressions Expressions containing SubexpressionVacancies.
         * Solutions are written after the last expression in `expressions`.
         * 
         * @param help_spaces Help space used in applying known integrals
         * @param applicability Result of `inclusive_scan` on `check_for_known_integrals()` applicability array
         */
        __global__ void
        apply_known_integrals(const ExpressionArray<SubexpressionCandidate> integrals,
                              ExpressionArray<> expressions, ExpressionArray<> help_spaces,
                              const Util::DeviceArray<uint32_t> applicability) {
            const size_t thread_count = Util::thread_count();
            const size_t thread_idx = Util::thread_idx();

            const size_t trans_step = thread_count / TRANSFORM_GROUP_SIZE;

            for (size_t trans_idx = thread_idx / TRANSFORM_GROUP_SIZE;
                 trans_idx < KnownIntegral::COUNT; trans_idx += trans_step) {
                for (size_t int_idx = thread_idx % TRANSFORM_GROUP_SIZE; int_idx < integrals.size();
                     int_idx += TRANSFORM_GROUP_SIZE) {
                    const size_t appl_idx = MAX_EXPRESSION_COUNT * trans_idx + int_idx;

                    if (!is_nonzero(appl_idx, applicability)) {
                        continue;
                    }

                    const size_t dest_idx =
                        expressions.size() + index_from_scan(applicability, appl_idx);

                    auto* const subexpr_candidate = expressions.at(dest_idx)
                                                    << SubexpressionCandidate::builder();
                    subexpr_candidate->copy_metadata_from(integrals[int_idx]);
                    KnownIntegral::APPLICATIONS[trans_idx](integrals[int_idx].arg().as<Integral>(),
                                                           subexpr_candidate->arg(),
                                                           help_spaces[dest_idx]);
                    subexpr_candidate->seal();

                    try_set_solver_idx(expressions, dest_idx);
                }
            }
        }

        /*
         * @brief Marks SubexpressionsVacancies as solved (sets `is_solved` and `solver_id`)
         * when there is a SubexpressionCandidate with all its SubexpressionVacancies solved.
         *
         * @param expressions Expressions to propagate information about being solved
         */
        __global__ void propagate_solved_subexpressions(ExpressionArray<> expressions) {
            const size_t thread_count = Util::thread_count();
            const size_t thread_idx = Util::thread_idx();

            // For each tree node there is a seperate starting thread.
            // If its node is solved it moves to it's parent.
            // It tries to fill the parent's vacancy with it's own solution.
            // If it succeeds and all of the parent's vacancies are solved, it moves to the parent.
            // This operation upwards is repeated upwards while all solutions to the current node exists
            // and the parent's vacancy is not solved and ends at the root.

            // Since `expr_idx = 0` is SubexpressionVacancy of the original integral, it is skipped
            for (size_t expr_idx = thread_idx + 1; expr_idx < expressions.size();
                 expr_idx += thread_count) {
                size_t current_expr_idx = expr_idx;
                while (current_expr_idx != 0) {
                    if (expressions[current_expr_idx].subexpression_candidate.subexpressions_left !=
                        0) {
                        break;
                    }

                    if (!try_set_solver_idx(expressions, current_expr_idx)) {
                        break;
                    }

                    // We iterate tree upwards.
                    // It may seem that there is a possibility of race condition
                    // when we will reach the same node, as the thread which has started the loop.
                    // However, since `try_set_solver_idx` is atomic, only one thread would be able
                    // to set `solver_idx` on the next parent and continue its journey upwards.
                    current_expr_idx = expressions[current_expr_idx]
                                           .subexpression_candidate.vacancy_expression_idx;
                }
            }
        }

        /*
         * @brief Finds redundant SubexpressionCandidates which are children of already solved SubexpressionVacancies.
         * SubexpressionCandidates that are solutions to SubexpressionsVacancies are not marked.
         *
         * @param expressions Expressions containing redundant SubexpressionCandidates
         * @param removability Solution. `0` is set for redundant SubexpresionCandidates
         */
        __global__ void find_redundand_expressions(const ExpressionArray<> expressions,
                                                   Util::DeviceArray<uint32_t> removability) {
            const size_t thread_count = Util::thread_count();
            const size_t thread_idx = Util::thread_idx();

            // Look further and further in the dependency tree and check whether we are not trying
            // to solve something that has been solved already
            for (size_t expr_idx = thread_idx; expr_idx < expressions.size();
                 expr_idx += thread_count) {
                removability[expr_idx] = 1;
                size_t current_expr_idx = expr_idx;

                while (current_expr_idx != 0) {
                    const size_t& parent_idx = expressions[current_expr_idx]
                                                   .subexpression_candidate.vacancy_expression_idx;
                    const size_t& parent_vacancy_idx =
                        expressions[current_expr_idx].subexpression_candidate.vacancy_idx;
                    const SubexpressionVacancy& parent_vacancy =
                        expressions[parent_idx][parent_vacancy_idx].subexpression_vacancy;

                    if (parent_vacancy.is_solved == 1 &&
                        parent_vacancy.solver_idx != current_expr_idx) {
                        removability[expr_idx] = 0;
                        break;
                    }

                    current_expr_idx = parent_idx;
                }
            }
        }

        /*
         * @brief Find integrals solving redundant SubexpressionVacancies
         *
         * @param integrals Integrals to be checked against
         * @param expressions Expressions pointing to integrals
         * @param expressions_removability Result of `find_redundand_expression()`. 
         * `0` for expressions to be deleted, `1` for the rest
         * @param integrals_removability Result. `0` for redundant integrals, 
         * `1` otherwise
         */
        __global__ void
        find_redundand_integrals(const ExpressionArray<> integrals,
                                 const ExpressionArray<> expressions,
                                 const Util::DeviceArray<uint32_t> expressions_removability,
                                 Util::DeviceArray<uint32_t> integrals_removability) {
            const size_t thread_count = Util::thread_count();
            const size_t thread_idx = Util::thread_idx();

            for (size_t int_idx = thread_idx; int_idx < integrals.size(); int_idx += thread_count) {
                const size_t& vacancy_expr_idx =
                    integrals[int_idx].subexpression_candidate.vacancy_expression_idx;
                const size_t& vacancy_idx = integrals[int_idx].subexpression_candidate.vacancy_idx;

                const bool parent_expr_failed = expressions_removability[vacancy_expr_idx] == 0;
                const bool parent_vacancy_solved =
                    expressions[vacancy_expr_idx][vacancy_idx].subexpression_vacancy.is_solved == 1;

                integrals_removability[int_idx] =
                    parent_expr_failed || parent_vacancy_solved ? 0 : 1;
            }
        }

        /*
         * @brief Moves `expressions` to `destinations` skipping those marked by `removability`.
         * Updates `solver_idx` and `vacancy_expression_idx`. Zeroes `candidate_integral_count`.
         *
         * @tparam ZERO_CANDIDATE_INTEGRAL_COUNT Whether to zero `candidate_integral_count` of
         * candidates that are moved to `destinations`
         * @param expressions Expressions to be moved
         * @param removability New locations indices of `expressions`. If `removability[i] == removability[i - 1]`
         * or `i == 0 && removability[i] != 0` then expression is moved to `destination[removability[i] - 1]`. 
         * @param destinations Destination to move integrals to
         */
        template <bool ZERO_CANDIDATE_INTEGRAL_COUNT = false>
        __global__ void remove_expressions(const ExpressionArray<> expressions,
                                           const Util::DeviceArray<uint32_t> removability,
                                           ExpressionArray<> destinations) {
            const size_t thread_count = Util::thread_count();
            const size_t thread_idx = Util::thread_idx();

            for (size_t expr_idx = thread_idx; expr_idx < expressions.size();
                 expr_idx += thread_count) {
                if (!is_nonzero(expr_idx, removability)) {
                    continue;
                }

                Symbol& destination = destinations[removability[expr_idx] - 1];
                expressions[expr_idx].copy_to(&destination);

                destination.if_is_do<SubexpressionCandidate>([&removability](auto& dst) {
                    dst.vacancy_expression_idx = removability[dst.vacancy_expression_idx] - 1;
                });

                for (size_t symbol_idx = 0; symbol_idx < destination.size(); ++symbol_idx) {
                    destination[symbol_idx].if_is_do<SubexpressionVacancy>(
                        [&removability](auto& vac) {
                            // We copy this value regardless of whether `vac` is really solved, if
                            // it is not, then `solver_idx` contains garbage anyways
                            vac.solver_idx = removability[vac.solver_idx] - 1;

                            if constexpr (ZERO_CANDIDATE_INTEGRAL_COUNT) {
                                vac.candidate_integral_count = 0;
                            }
                        });
                }
            }
        }

        /*
         * @brief Moves `integrals` to `destinations` removing those specified by
         * `removability` and updates `vacancy_expression_idx`.
         *
         * @param integrals Integrals to be moved
         * @param integrals_removability Indexes of integrals in `destinations`.
         * When `integrals_removability[i] == integrals_removability[i - 1]`
         * or `i == 0 && removability[i] != 0` the expression is moved to
         * `destinations[removability[i] - 1]`.
         * @param destinations Place to move correct integrals to
         */
        __global__ void remove_integrals(const ExpressionArray<SubexpressionCandidate> integrals,
                                         const Util::DeviceArray<uint32_t> integrals_removability,
                                         const Util::DeviceArray<uint32_t> expressions_removability,
                                         ExpressionArray<> destinations) {
            const size_t thread_count = Util::thread_count();
            const size_t thread_idx = Util::thread_idx();

            for (size_t int_idx = thread_idx; int_idx < integrals.size(); int_idx += thread_count) {
                if (!is_nonzero(int_idx, integrals_removability)) {
                    continue;
                }

                Symbol& destination = destinations[integrals_removability[int_idx] - 1];
                integrals[int_idx].symbol()->copy_to(&destination);

                size_t& vacancy_expr_idx =
                    destination.as<SubexpressionCandidate>().vacancy_expression_idx;
                vacancy_expr_idx = expressions_removability[vacancy_expr_idx] - 1;
            }
        }

        /*
         * @brief Checks which heuristics are applicable to which integrals and updates
         * `candidate_integral_count` and `candidate_expression_count` in correct expressions 
         *
         * @param integrals Integrals to be checked
         * @param expressions Parents of SubexpressionCandidate in `integrals`.
         * @param new_integrals_flags Solution. When `integrals[i]` matches `Heuristic::CHECKS[j]`
         * sets `new_integrals_flags[MAX_EXPRESSION_COUNT * j + i]` to `1`, otherwise `0`. 
         * @param new_expressions_flags  Solution. When `integrals[i]` matches `Heuristic::CHECKS[j]`
         * sets `new_expressions_flags[MAX_EXPRESSION_COUNT * j + i]` to `1`, otherwise `0`. 
         */
        __global__ void
        check_heuristics_applicability(const ExpressionArray<SubexpressionCandidate> integrals,
                                       ExpressionArray<> expressions,
                                       Util::DeviceArray<uint32_t> new_integrals_flags,
                                       Util::DeviceArray<uint32_t> new_expressions_flags) {
            const size_t thread_count = Util::thread_count();
            const size_t thread_idx = Util::thread_idx();

            const size_t check_step = thread_count / TRANSFORM_GROUP_SIZE;

            for (size_t check_idx = thread_idx / TRANSFORM_GROUP_SIZE; check_idx < Heuristic::COUNT;
                 check_idx += check_step) {
                for (size_t int_idx = thread_idx % TRANSFORM_GROUP_SIZE; int_idx < integrals.size();
                     int_idx += TRANSFORM_GROUP_SIZE) {
                    size_t appl_idx = MAX_EXPRESSION_COUNT * check_idx + int_idx;
                    Heuristic::CheckResult result =
                        Heuristic::CHECKS[check_idx](integrals[int_idx].arg().as<Integral>());
                    new_integrals_flags[appl_idx] = result.new_integrals;
                    new_expressions_flags[appl_idx] = result.new_expressions;

                    const size_t& vacancy_expr_idx = integrals[int_idx].vacancy_expression_idx;
                    const size_t& vacancy_idx = integrals[int_idx].vacancy_idx;
                    SubexpressionVacancy& parent_vacancy =
                        expressions[vacancy_expr_idx][vacancy_idx].subexpression_vacancy;

                    if (result.new_expressions == 0) {
                        // Assume new integrals are direct children of the vacancy
                        atomicAdd(&parent_vacancy.candidate_integral_count, result.new_integrals);
                    }
                    else {
                        // Assume new integrals are going to be children of new expressions, which
                        // are going to be children of the vacancy
                        atomicAdd(&parent_vacancy.candidate_expression_count,
                                  result.new_expressions);
                    }
                }
            }
        }

        /*
         * @brief Applies heuristics to integrals
         *
         * @param integrals Integrals on which heuristics will be applied
         * @param integrals_destinations Solutions destination
         * @param expressions_destinations Destination for new expressions. 
         * New expressions will be appended to already existing ones.
         * @param help_spaces Help space for transformations
         * @param new_integrals_indices Indices of new integrals incremented by 1.
         * If given index is equal to its predecessor, then its integral and heuristic
         * (specified in `check_heuristics_applicability()`) haven't found any solution.
         * `new_integrals_indices[0]` will override `1` to `0`.
         * @param new_expressions_indices Analogical to `new_integrals_indices` for `expressions`
         */
        __global__ void
        apply_heuristics(const ExpressionArray<SubexpressionCandidate> integrals,
                         ExpressionArray<> integrals_destinations,
                         ExpressionArray<> expressions_destinations, ExpressionArray<> help_spaces,
                         const Util::DeviceArray<uint32_t> new_integrals_indices,
                         const Util::DeviceArray<uint32_t> new_expressions_indices) {
            const size_t thread_count = Util::thread_count();
            const size_t thread_idx = Util::thread_idx();

            const size_t trans_step = thread_count / TRANSFORM_GROUP_SIZE;

            for (size_t trans_idx = thread_idx / TRANSFORM_GROUP_SIZE; trans_idx < Heuristic::COUNT;
                 trans_idx += trans_step) {
                for (size_t int_idx = thread_idx % TRANSFORM_GROUP_SIZE; int_idx < integrals.size();
                     int_idx += TRANSFORM_GROUP_SIZE) {
                    const size_t appl_idx = MAX_EXPRESSION_COUNT * trans_idx + int_idx;
                    if (!is_nonzero(appl_idx, new_integrals_indices)) {
                        continue;
                    }

                    const size_t int_dst_idx = index_from_scan(new_integrals_indices, appl_idx);

                    if (new_expressions_indices[appl_idx] != 0) {
                        const size_t expr_dst_idx =
                            expressions_destinations.size() +
                            index_from_scan(new_expressions_indices, appl_idx);
                        Heuristic::APPLICATIONS[trans_idx](
                            integrals[int_idx], integrals_destinations.iterator(int_dst_idx),
                            expressions_destinations.iterator(expr_dst_idx),
                            help_spaces[int_dst_idx]);
                    }
                    else {
                        Heuristic::APPLICATIONS[trans_idx](
                            integrals[int_idx], integrals_destinations.iterator(int_dst_idx),
                            ExpressionArray<>::Iterator::null(), help_spaces[int_dst_idx]);
                    }
                }
            }
        }

        /*
         * @brief Propagates information about failed SubexpressionVacancy upwards to parent
         * expressions.
         *
         * @param expressions Expressions to update
         * @param failures Array that should be filled with values of `1`, if `expressions[i]` fails
         * then `failures[i]` is set to 0
         */
        __global__ void propagate_failures_upwards(ExpressionArray<> expressions,
                                                   Util::DeviceArray<uint32_t> failures) {
            const size_t thread_count = Util::thread_count();
            const size_t thread_idx = Util::thread_idx();

            for (size_t expr_idx = thread_idx; expr_idx < expressions.size();
                 expr_idx += thread_count) {
                SubexpressionCandidate& self_candidate =
                    expressions[expr_idx].subexpression_candidate;

                // Some other thread was here already, as `failures` starts with 1 everywhere
                if (failures[expr_idx] == 0) {
                    continue;
                }

                bool is_failed = false;

                // expressions[current_expr_idx][0] is subexpression_candidate, so it could be
                // skipped, but if `expr_idx == 0` it is the only SubexpressionVacancy
                for (size_t sym_idx = 0; sym_idx < expressions[expr_idx].size(); ++sym_idx) {
                    if (!expressions[expr_idx][sym_idx].is(Type::SubexpressionVacancy)) {
                        continue;
                    }

                    SubexpressionVacancy& vacancy =
                        expressions[expr_idx][sym_idx].subexpression_vacancy;

                    if (vacancy.candidate_integral_count == 0 &&
                        vacancy.candidate_expression_count == 0 && vacancy.is_solved == 0) {
                        is_failed = true;
                        break;
                    }
                }

                if (!is_failed || !try_set(failures[expr_idx], 0U)) {
                    continue;
                }

                size_t current_expr_idx = expr_idx;
                while (current_expr_idx != 0) {
                    const size_t& parent_idx = expressions[current_expr_idx]
                                                   .subexpression_candidate.vacancy_expression_idx;
                    const size_t& vacancy_idx =
                        expressions[current_expr_idx].subexpression_candidate.vacancy_idx;
                    SubexpressionVacancy& parent_vacancy =
                        expressions[parent_idx][vacancy_idx].subexpression_vacancy;

                    if (parent_vacancy.candidate_integral_count != 0 ||
                        parent_vacancy.is_solved == 1) {
                        break;
                    }

                    const size_t parent_vacancy_candidates_left =
                        atomicSub(&parent_vacancy.candidate_expression_count, 1) - 1;

                    // Go upwards if parent is failed
                    if (parent_vacancy_candidates_left != 0 || !try_set(failures[parent_idx], 0U)) {
                        break;
                    }

                    current_expr_idx = parent_idx;
                }
            }
        }

        /*
         * @brief Propagates information about failed SubexpressionCandidate downwards
         *
         * @param expression Expressions to update
         * @param failurs Arrays that should point out already failed expressions, all descendands
         * of which are going to be failed (failures[i] == 0 iff failed, 1 otherwise)
         */
        __global__ void propagate_failures_downwards(ExpressionArray<> expressions,
                                                     Util::DeviceArray<uint32_t> failures) {
            const size_t thread_count = Util::thread_count();
            const size_t thread_idx = Util::thread_idx();

            // Top expression has no parents, so we skip it
            for (size_t expr_idx = thread_idx + 1; expr_idx < expressions.size();
                 expr_idx += thread_count) {
                size_t current_expr_idx = expr_idx;

                while (current_expr_idx != 0) {
                    const size_t& parent_idx = expressions[current_expr_idx]
                                                   .subexpression_candidate.vacancy_expression_idx;

                    if (failures[parent_idx] == 0) {
                        failures[expr_idx] = 0;
                        break;
                    }

                    current_expr_idx = parent_idx;
                }
            }
        }

        /*
         * @brief Marks integrals that point to expressions which are going to be removed
         *
         * @param integrals Integrals to mark
         * @param expressions_removability Expressions which are going to be removed, 0 for the ones
         * awaiting removal, 1 for the ones staying
         * @param integrals_removability Checking result, same convention as in
         * `expressions_removability`
         */
        __global__ void
        find_redundand_integrals(const ExpressionArray<> integrals,
                                 const Util::DeviceArray<uint32_t> expressions_removability,
                                 Util::DeviceArray<uint32_t> integrals_removability) {
            const size_t thread_count = Util::thread_count();
            const size_t thread_idx = Util::thread_idx();

            for (size_t int_idx = thread_idx; int_idx < integrals.size(); int_idx += thread_count) {
                const size_t& parent_idx =
                    integrals[int_idx].subexpression_candidate.vacancy_expression_idx;

                integrals_removability[int_idx] = expressions_removability[parent_idx];
            }
        }

        /*
         * @brief Replaces nth symbol in `expression` with `tree`, skipping the first element of
         * `tree` and expanding substitutions if `Solution` is the second symbol in `tree`
         *
         * @param expression Expression to make the replacement in
         * @param n Index of symbol to replace
         * @param tree Expression to make replacement with. Its first symbol is skipped (assumed to
         * be SubexpressionCandidate)
         *
         * @return Copy of `expression` with the replacement
         */
        std::vector<Sym::Symbol> replace_nth_with_tree(std::vector<Sym::Symbol> expression,
                                                       const size_t n,
                                                       const std::vector<Sym::Symbol>& tree) {
            if constexpr (Consts::DEBUG) {
                if (!tree[0].is(Type::SubexpressionCandidate)) {
                    Util::crash(
                        "Invalid first symbol of tree: %s, should be SubexpressionCandidate",
                        type_name(tree[0].type()));
                }
            }

            std::vector<Sym::Symbol> tree_content;

            if (tree[1].is(Sym::Type::Solution)) {
                tree_content = tree[1].as<Sym::Solution>().substitute_substitutions();
            }
            else {
                tree_content.resize(tree.size() - 1);
                std::copy(tree.begin() + 1, tree.end(), tree_content.begin());
            }

            expression[n].init_from(Sym::ExpanderPlaceholder::with_size(tree_content.size()));

            std::vector<Sym::Symbol> new_tree(expression.size() + tree_content.size() - 1);
            expression.data()->compress_to(*new_tree.data());

            std::copy(tree_content.begin(), tree_content.end(),
                      new_tree.begin() + static_cast<int64_t>(n));

            return new_tree;
        }

        /*
         * @brief Collapses a tree of expressions with Solutions with Substitutions and
         * interreferencing SubexpressionCandidates and SubexpressionVacancies to a single
         * expression.
         *
         * @param tree Tree to collapse
         * @param n Index of tree node serving as tree root
         *
         * @return Collapsed tree
         */
        std::vector<Sym::Symbol> collapse_nth(const std::vector<std::vector<Sym::Symbol>>& tree,
                                              const size_t n) {
            std::vector<Sym::Symbol> current_collapse = tree[n];

            for (size_t i = 0; i < current_collapse.size(); ++i) {
                if (!current_collapse[i].is(Sym::Type::SubexpressionVacancy)) {
                    continue;
                }

                const auto subtree = collapse_nth(
                    tree, current_collapse[i].as<Sym::SubexpressionVacancy>().solver_idx);

                auto new_collapse = replace_nth_with_tree(current_collapse, i, subtree);
                i += new_collapse.size() - current_collapse.size();
                current_collapse = new_collapse;
            }

            return current_collapse;
        }

        /*
         * @brief Collapses a tree of expressions with Solutions with Substitutions and
         * interreferencing SubexpressionCandidates and SubexpressionVacancies to a single
         * expression
         *
         * @param tree Tree to collapse
         *
         * @return Collapsed tree
         */
        std::vector<Sym::Symbol> collapse(const std::vector<std::vector<Sym::Symbol>>& tree) {
            auto collapsed = collapse_nth(tree, 0);
            std::vector<Sym::Symbol> reversed(collapsed.size());
            const size_t new_size = collapsed.data()->compress_reverse_to(reversed.data());
            Sym::Symbol::copy_and_reverse_symbol_sequence(collapsed.data(), reversed.data(),
                                                          new_size);

            std::vector<Sym::Symbol> help_space(EXPRESSION_MAX_SYMBOL_COUNT);
            collapsed.data()->simplify(help_space.data());
            collapsed.resize(collapsed.data()->size());

            return collapsed;
        }
    }

    std::optional<std::vector<Symbol>> solve_integral(const std::vector<Symbol>& integral) {
        static constexpr size_t BLOCK_SIZE = 512;
        static constexpr size_t BLOCK_COUNT = 32;
        const size_t MAX_CHECK_COUNT =
            KnownIntegral::COUNT > Heuristic::COUNT ? KnownIntegral::COUNT : Heuristic::COUNT;
        const size_t SCAN_ARRAY_SIZE = MAX_CHECK_COUNT * MAX_EXPRESSION_COUNT;

        ExpressionArray<> expressions({single_integral_vacancy()}, EXPRESSION_MAX_SYMBOL_COUNT,
                                      MAX_EXPRESSION_COUNT);
        ExpressionArray<> expressions_swap(MAX_EXPRESSION_COUNT, EXPRESSION_MAX_SYMBOL_COUNT,
                                           expressions.size());

        ExpressionArray<SubexpressionCandidate> integrals({first_expression_candidate(integral)},
                                                          MAX_EXPRESSION_COUNT,
                                                          EXPRESSION_MAX_SYMBOL_COUNT);
        ExpressionArray<SubexpressionCandidate> integrals_swap(MAX_EXPRESSION_COUNT,
                                                               EXPRESSION_MAX_SYMBOL_COUNT);
        ExpressionArray<> help_spaces(MAX_EXPRESSION_COUNT, EXPRESSION_MAX_SYMBOL_COUNT,
                                      integrals.size());
        Util::DeviceArray<uint32_t> scan_array_1(SCAN_ARRAY_SIZE, true);
        Util::DeviceArray<uint32_t> scan_array_2(SCAN_ARRAY_SIZE, true);

        for (size_t i = 0;; ++i) {
            simplify<<<BLOCK_COUNT, BLOCK_SIZE>>>(integrals, help_spaces);
            hipDeviceSynchronize();

            check_for_known_integrals<<<BLOCK_COUNT, BLOCK_SIZE>>>(integrals, scan_array_1);
            hipDeviceSynchronize();

            thrust::inclusive_scan(thrust::device, scan_array_1.begin(), scan_array_1.end(),
                                   scan_array_1.data());
            hipDeviceSynchronize();

            apply_known_integrals<<<BLOCK_COUNT, BLOCK_SIZE>>>(integrals, expressions, help_spaces,
                                                               scan_array_1);
            hipDeviceSynchronize();
            expressions.increment_size_from_device(scan_array_1.last());

            propagate_solved_subexpressions<<<BLOCK_COUNT, BLOCK_SIZE>>>(expressions);
            hipDeviceSynchronize();

            std::vector<Symbol> first_expression = expressions.to_vector(0);
            if (first_expression.data()->as<SubexpressionVacancy>().is_solved == 1) {
                return collapse(expressions.to_vector());
            }

            scan_array_1.zero_mem();
            find_redundand_expressions<<<BLOCK_COUNT, BLOCK_SIZE>>>(expressions, scan_array_1);
            hipDeviceSynchronize();

            scan_array_2.zero_mem(); // TODO: Not necessary?
            find_redundand_integrals<<<BLOCK_COUNT, BLOCK_SIZE>>>(integrals, expressions,
                                                                  scan_array_1, scan_array_2);
            hipDeviceSynchronize();

            thrust::inclusive_scan(thrust::device, scan_array_1.begin(), scan_array_1.end(),
                                   scan_array_1.data());
            thrust::inclusive_scan(thrust::device, scan_array_2.begin(), scan_array_2.end(),
                                   scan_array_2.data());
            hipDeviceSynchronize();

            remove_expressions<true>
                <<<BLOCK_COUNT, BLOCK_SIZE>>>(expressions, scan_array_1, expressions_swap);
            remove_integrals<<<BLOCK_COUNT, BLOCK_SIZE>>>(integrals, scan_array_2, scan_array_1,
                                                          integrals_swap);
            hipDeviceSynchronize();

            std::swap(expressions, expressions_swap);
            std::swap(integrals, integrals_swap);
            expressions.resize_from_device(scan_array_1.last());
            integrals.resize_from_device(scan_array_2.last());

            scan_array_1.zero_mem();
            scan_array_2.zero_mem();
            check_heuristics_applicability<<<BLOCK_COUNT, BLOCK_SIZE>>>(integrals, expressions,
                                                                        scan_array_1, scan_array_2);
            hipDeviceSynchronize();

            thrust::inclusive_scan(thrust::device, scan_array_1.begin(), scan_array_1.end(),
                                   scan_array_1.data());
            thrust::inclusive_scan(thrust::device, scan_array_2.begin(), scan_array_2.end(),
                                   scan_array_2.data());
            hipDeviceSynchronize();

            apply_heuristics<<<BLOCK_COUNT, BLOCK_SIZE>>>(integrals, integrals_swap, expressions,
                                                          help_spaces, scan_array_1, scan_array_2);
            hipDeviceSynchronize();

            std::swap(integrals, integrals_swap);
            integrals.resize_from_device(scan_array_1.last());
            expressions.increment_size_from_device(scan_array_2.last());

            scan_array_1.set_mem(1);
            hipDeviceSynchronize();

            propagate_failures_upwards<<<BLOCK_COUNT, BLOCK_SIZE>>>(expressions, scan_array_1);
            hipDeviceSynchronize();

            // First expression in the array has failed, all is lost
            if (scan_array_1.to_cpu(0) == 0) {
                return std::nullopt;
            }

            propagate_failures_downwards<<<BLOCK_COUNT, BLOCK_SIZE>>>(expressions, scan_array_1);
            hipDeviceSynchronize();

            scan_array_2.zero_mem();
            find_redundand_integrals<<<BLOCK_COUNT, BLOCK_SIZE>>>(integrals, scan_array_1,
                                                                  scan_array_2);
            hipDeviceSynchronize();

            thrust::inclusive_scan(thrust::device, scan_array_1.begin(), scan_array_1.end(),
                                   scan_array_1.data());
            thrust::inclusive_scan(thrust::device, scan_array_2.begin(), scan_array_2.end(),
                                   scan_array_2.data());
            hipDeviceSynchronize();

            remove_expressions<false>
                <<<BLOCK_COUNT, BLOCK_SIZE>>>(expressions, scan_array_1, expressions_swap);
            remove_integrals<<<BLOCK_COUNT, BLOCK_SIZE>>>(integrals, scan_array_2, scan_array_1,
                                                          integrals_swap);
            hipDeviceSynchronize();

            std::swap(expressions, expressions_swap);
            std::swap(integrals, integrals_swap);

            // How many expressions are left, cannot take scan_array_1.last() because space
            // after last place in scan_array_1 that corresponds to an expression is occupied
            // by ones
            expressions.resize(scan_array_1.to_cpu(expressions_swap.size() - 1));
            integrals.resize_from_device(scan_array_2.last());
            hipDeviceSynchronize();

            scan_array_1.zero_mem();
            scan_array_2.zero_mem();
        }

        return std::nullopt;
    }
}

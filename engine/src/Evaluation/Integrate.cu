#include "hip/hip_runtime.h"
#include "Integrate.cuh"

#include "Utils/Cuda.cuh"

namespace {
    __device__ bool is_zero_size(const size_t index,
                                 const Util::DeviceArray<size_t>& inclusive_scan) {
        return index == 0 && inclusive_scan[index] != 0 ||
               index != 0 && inclusive_scan[index - 1] != inclusive_scan[index];
    }
}

namespace Sym {
    __device__ const ApplicabilityCheck known_integral_checks[] = {
        is_single_variable, is_simple_variable_power, is_variable_exponent,
        is_simple_sine,     is_simple_cosine,         is_constant,
        is_known_arctan};

    __device__ const IntegralTransform known_integral_applications[] = {
        integrate_single_variable, integrate_simple_variable_power, integrate_variable_exponent,
        integrate_simple_sine,     integrate_simple_cosine,         integrate_constant,
        integrate_arctan};

    static_assert(sizeof(known_integral_applications) == sizeof(known_integral_checks),
                  "Different number of heuristics and applications defined");

    static_assert(sizeof(known_integral_checks) ==
                      sizeof(ApplicabilityCheck) * KNOWN_INTEGRAL_COUNT,
                  "HEURISTIC_CHECK_COUNT is not equal to number of heuristic checks");

    __device__ const ApplicabilityCheck heuristic_checks[] = {is_function_of_ex};

    __device__ const IntegralTransform heuristic_applications[] = {transform_function_of_ex};

    static_assert(sizeof(heuristic_checks) == sizeof(heuristic_applications),
                  "Different number of heuristics and applications defined");

    static_assert(sizeof(heuristic_checks) == sizeof(ApplicabilityCheck) * HEURISTIC_CHECK_COUNT,
                  "HEURISTIC_CHECK_COUNT is not equal to number of heuristic checks");

    __device__ Symbol ex_function[3];
    __device__ void init_ex_function() {
        Power* const power = ex_function << Power::builder();
        power->arg1().known_constant = KnownConstant::with_value(KnownConstantValue::E);
        power->seal_arg1();
        power->arg2().variable = Variable::create();
        power->seal();
    }

    __device__ size_t is_function_of_ex(const Integral* const integral) {
        // TODO: Move somewhere so that it's initialized only once and not every time this function
        // is called
        init_ex_function();
        return integral->integrand()->is_function_of(ex_function) ? 1 : 0;
    }

    __device__ void transform_function_of_ex(const Integral* const integral,
                                             Symbol* const destination, Symbol* const help_space) {
        // TODO: Move somewhere so that it's initialized only once and not every time this function
        // is called
        init_ex_function();
        Symbol variable{};
        variable.variable = Variable::create();

        integral->integrate_by_substitution_with_derivative(ex_function, &variable, destination,
                                                            help_space);
    }

    __device__ size_t is_single_variable(const Integral* const integral) {
        return integral->integrand()->is(Type::Variable) ? 1 : 0;
    }

    __device__ size_t is_simple_variable_power(const Integral* const integral) {
        const Symbol* const integrand = integral->integrand();
        if (!integrand[0].is(Type::Power) || !integrand[1].is(Type::Variable)) {
            return 0;
        }

        if (integrand[2].is(Type::NumericConstant) && integrand[2].numeric_constant.value == -1.0) {
            return 0;
        }

        return integrand[2].is_constant() ? 0 : 1;
    }
    __device__ size_t is_variable_exponent(const Integral* const integral) {
        const Symbol* const integrand = integral->integrand();
        return integrand[0].is(Type::Power) && integrand[1].is(Type::KnownConstant) &&
                       integrand[1].known_constant.value == KnownConstantValue::E &&
                       integrand[2].is(Type::Variable)
                   ? 1
                   : 0;
    }
    __device__ size_t is_simple_sine(const Integral* const integral) {
        const Symbol* const integrand = integral->integrand();
        return integrand[0].is(Type::Sine) && integrand[1].is(Type::Variable) ? 1 : 0;
    }

    __device__ size_t is_simple_cosine(const Integral* const integral) {
        const Symbol* const integrand = integral->integrand();
        return integrand[0].is(Type::Cosine) && integrand[1].is(Type::Variable) ? 1 : 0;
    }

    __device__ size_t is_constant(const Integral* const integral) {
        const Symbol* const integrand = integral->integrand();
        return integrand->is_constant() ? 1 : 0;
    }

    __device__ size_t is_known_arctan(const Integral* const integral) {
        const Symbol* const integrand = integral->integrand();
        // 1/(x^2+1) or 1/(1+x^2)
        return integrand[0].is(Type::Product) && integrand[1].is(Type::NumericConstant) &&
                       integrand[1].numeric_constant.value == 1.0 &&
                       integrand[2].is(Type::Reciprocal) && integrand[3].is(Type::Addition) &&
                       ((integrand[4].is(Type::Power) && integrand[5].is(Type::Variable) &&
                         integrand[6].is(Type::NumericConstant) &&
                         integrand[6].numeric_constant.value == 2.0 &&
                         integrand[7].is(Type::NumericConstant) &&
                         integrand[7].numeric_constant.value == 1.0) ||
                        (integrand[4].is(Type::NumericConstant) &&
                         integrand[4].numeric_constant.value == 1.0 &&
                         integrand[5].is(Type::Power) && integrand[6].is(Type::Variable) &&
                         integrand[7].is(Type::NumericConstant) &&
                         integrand[7].numeric_constant.value == 2.0))
                   ? 1
                   : 0;
    }

    __device__ void integrate_single_variable(const Integral* const integral,
                                              Symbol* const destination,
                                              Symbol* const /*help_space*/) {

        Symbol* const solution_expr = prepare_solution(integral, destination);

        Product* const product = solution_expr << Product::builder();
        product->arg1().numeric_constant = NumericConstant::with_value(0.5);
        product->seal_arg1();

        Power* const power = product->arg2() << Power::builder();
        power->arg1().variable = Variable::create();
        power->seal_arg1();
        power->arg2().numeric_constant = NumericConstant::with_value(2.0);
        power->seal();
        product->seal();

        destination->solution.seal();
    }

    __device__ void integrate_simple_variable_power(const Integral* const integral,
                                                    Symbol* const destination,
                                                    Symbol* const /*help_space*/) {
        const Symbol* const integrand = integral->integrand();

        Symbol* const solution_expr = prepare_solution(integral, destination);
        const Symbol* const exponent = &integral->integrand()->power.arg2();

        // 1/(c+1) * x^(c+1), c może być całym drzewem
        Product* const product = solution_expr << Product::builder();

        Reciprocal* const reciprocal = product->arg1() << Reciprocal::builder();
        Addition* const multiplier_addition = reciprocal->arg() << Addition::builder();
        exponent->copy_to(&multiplier_addition->arg1());
        multiplier_addition->seal_arg1();
        multiplier_addition->arg2().numeric_constant = NumericConstant::with_value(1.0);
        multiplier_addition->seal();
        reciprocal->seal();
        product->seal_arg1();

        Power* const power = product->arg2() << Power::builder();
        power->arg1().variable = Variable::create();
        power->seal_arg1();
        Addition* const exponent_addition = power->arg2() << Addition::builder();
        exponent->copy_to(&exponent_addition->arg1());
        exponent_addition->seal_arg1();
        exponent_addition->arg2().numeric_constant = NumericConstant::with_value(1.0);
        exponent_addition->seal();
        power->seal();
        product->seal();

        destination->solution.seal();
    }

    __device__ void integrate_variable_exponent(const Integral* const integral,
                                                Symbol* const destination,
                                                Symbol* const /*help_space*/) {
        Symbol* const solution_expr = prepare_solution(integral, destination);
        const Symbol* const integrand = integral->integrand();

        Power* const power = solution_expr << Power::builder();
        power->arg1().known_constant = KnownConstant::with_value(KnownConstantValue::E);
        power->seal_arg1();
        power->arg2().variable = Variable::create();
        power->seal();

        destination->solution.seal();
    }

    __device__ void integrate_simple_sine(const Integral* const integral, Symbol* const destination,
                                          Symbol* const /*help_space*/) {
        Symbol* const solution_expr = prepare_solution(integral, destination);
        const Symbol* const integrand = integral->integrand();

        Negation* const minus = solution_expr << Negation::builder();
        Cosine* const cos = minus->arg() << Cosine::builder();
        cos->arg().variable = Variable::create();
        cos->seal();
        minus->seal();

        destination->solution.seal();
    }

    __device__ void integrate_simple_cosine(const Integral* const integral,
                                            Symbol* const destination,
                                            Symbol* const /*help_space*/) {
        Symbol* const solution_expr = prepare_solution(integral, destination);
        const Symbol* const integrand = integral->integrand();

        Sine* const sine = solution_expr << Sine::builder();
        sine->arg().variable = Variable::create();
        sine->seal();

        destination->solution.seal();
    }

    __device__ void integrate_constant(const Integral* const integral, Symbol* const destination,
                                       Symbol* const /*help_space*/) {
        const Symbol* const integrand = integral->integrand();
        Symbol* const solution_expr = prepare_solution(integral, destination);

        Product* const product = solution_expr << Product::builder();
        product->arg1().variable = Variable::create();
        product->seal_arg1();
        integrand->copy_to(&product->arg2());
        product->seal();

        destination->solution.seal();
    }

    __device__ void integrate_arctan(const Integral* const integral, Symbol* const destination,
                                     Symbol* const /*help_space*/) {
        const Symbol* const integrand = integral->integrand();
        Symbol* const solution_expr = prepare_solution(integral, destination);

        Arctangent* const arctangent = solution_expr << Arctangent::builder();
        arctangent->arg().variable = Variable::create();
        arctangent->seal();

        destination->solution.seal();
    }

    __device__ Symbol* prepare_solution(const Integral* const integral, Symbol* const destination) {
        Solution* const solution = destination << Solution::builder();
        Symbol::copy_symbol_sequence(Symbol::from(solution->first_substitution()),
                                     Symbol::from(integral->first_substitution()),
                                     integral->substitutions_size());
        solution->seal_substitutions(integral->substitution_count, integral->substitutions_size());

        return solution->expression();
    }

    __device__ void check_applicability(const ExpressionArray<Integral>& integrals,
                                        Util::DeviceArray<size_t>& applicability,
                                        const ApplicabilityCheck* const checks,
                                        const size_t check_count) {
        const size_t thread_count = Util::thread_count();
        const size_t thread_idx = Util::thread_idx();

        const size_t check_step = thread_count / TRANSFORM_GROUP_SIZE;

        for (size_t check_idx = thread_idx / TRANSFORM_GROUP_SIZE; check_idx < check_count;
             check_idx += check_step) {
            for (size_t int_idx = thread_idx % TRANSFORM_GROUP_SIZE; int_idx < integrals.size();
                 int_idx += TRANSFORM_GROUP_SIZE) {
                size_t appl_idx = MAX_INTEGRAL_COUNT * check_idx + int_idx;
                applicability[appl_idx] = checks[check_idx](integrals[int_idx]);
            }
        }
    }

    __device__ void
    apply_transforms(const ExpressionArray<Integral>& integrals, ExpressionArray<>& destinations,
                     ExpressionArray<>& help_spaces, const Util::DeviceArray<size_t>& applicability,
                     const IntegralTransform* const transforms, const size_t transform_count) {
        const size_t thread_count = Util::thread_count();
        const size_t thread_idx = Util::thread_idx();

        const size_t trans_step = thread_count / TRANSFORM_GROUP_SIZE;

        for (size_t trans_idx = thread_idx / TRANSFORM_GROUP_SIZE; trans_idx < transform_count;
             trans_idx += trans_step) {
            for (size_t int_idx = thread_idx % TRANSFORM_GROUP_SIZE; int_idx < integrals.size();
                 int_idx += TRANSFORM_GROUP_SIZE) {
                const size_t appl_index = MAX_INTEGRAL_COUNT * trans_idx + int_idx;
                if (is_zero_size(appl_index, applicability)) {
                    const size_t dest_idx = applicability[appl_index] - 1;
                    transforms[trans_idx](integrals[int_idx], destinations[dest_idx],
                                          help_spaces[dest_idx]);
                }
            }
        }
    }

    __global__ void check_for_known_integrals(const ExpressionArray<Integral> integrals,
                                              Util::DeviceArray<size_t> applicability) {
        check_applicability(integrals, applicability, known_integral_checks, KNOWN_INTEGRAL_COUNT);
    }

    __global__ void apply_known_integrals(const ExpressionArray<Integral> integrals,
                                          ExpressionArray<> destinations,
                                          ExpressionArray<> help_spaces,
                                          const Util::DeviceArray<size_t> applicability) {
        apply_transforms(integrals, destinations, help_spaces, applicability,
                         known_integral_applications, KNOWN_INTEGRAL_COUNT);
    }

    __global__ void check_heuristics_applicability(const ExpressionArray<Integral> integrals,
                                                   Util::DeviceArray<size_t> applicability) {
        check_applicability(integrals, applicability, heuristic_checks, HEURISTIC_CHECK_COUNT);
    }

    __global__ void apply_heuristics(const ExpressionArray<Integral> integrals,
                                     ExpressionArray<> destinations, ExpressionArray<> help_spaces,
                                     const Util::DeviceArray<size_t> applicability) {
        apply_transforms(integrals, destinations, help_spaces, applicability,
                         heuristic_applications, HEURISTIC_CHECK_COUNT);
    }

    __global__ void simplify(ExpressionArray<> expressions, ExpressionArray<> help_spaces) {
        const size_t thread_count = Util::thread_count();
        const size_t thread_idx = Util::thread_idx();

        for (size_t expr_idx = thread_idx; expr_idx < expressions.size();
             expr_idx += thread_count) {
            expressions[expr_idx]->simplify(help_spaces[expr_idx]);
        }
    }
}

#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstring>

#include <iostream>
#include <vector>

#include <thrust/execution_policy.h>
#include <thrust/scan.h>

#include "integral.cuh"
#include "integrate.cuh"
#include "symbol.cuh"

static constexpr size_t BLOCK_SIZE = 128;
static constexpr size_t BLOCK_COUNT = 32;

int main() {
    std::vector<Sym::Symbol> ixpr = Sym::integral(Sym::var() ^ Sym::num(2));
    std::cout << "ixpr1: " << ixpr[0].to_string() << std::endl;

    std::vector<Sym::Symbol> ixpr2 = Sym::substitute(ixpr, Sym::cos(Sym::var()));
    std::cout << "ixpr2: " << ixpr2[0].to_string() << std::endl;

    std::vector<Sym::Symbol> ixpr3 = Sym::substitute(ixpr2, Sym::e() ^ Sym::var());
    std::cout << "ixpr3: " << ixpr3[0].to_string() << std::endl;

    std::cout << "Creating an expression" << std::endl;
    std::vector<std::vector<Sym::Symbol>> expressions = {Sym::cos(Sym::var()),
                                                         Sym::sin(Sym::cos(Sym::var())),
                                                         Sym::e() ^ Sym::var(),
                                                         Sym::var() ^ Sym::num(5),
                                                         Sym::var() ^ Sym::pi(),
                                                         Sym::var() ^ Sym::var(),
                                                         Sym::pi()};

    for (size_t i = 0; i < expressions.size(); ++i) {
        std::cout << expressions[i][0].to_string() << std::endl;
    }

    std::cout << "Allocating and zeroing GPU memory" << std::endl;

    size_t mem_total = 0;

    Sym::Symbol* d_expressions;
    hipMalloc(&d_expressions, Sym::EXPRESSION_ARRAY_SIZE * sizeof(Sym::Symbol));
    hipMemset(d_expressions, 0, Sym::EXPRESSION_ARRAY_SIZE * sizeof(Sym::Symbol));
    mem_total += Sym::EXPRESSION_ARRAY_SIZE * sizeof(Sym::Symbol);

    Sym::Symbol* d_expressions_swap;
    hipMalloc(&d_expressions_swap, Sym::EXPRESSION_ARRAY_SIZE * sizeof(Sym::Symbol));
    mem_total += Sym::EXPRESSION_ARRAY_SIZE * sizeof(Sym::Symbol);

    size_t* d_applicability;
    hipMalloc(&d_applicability, Sym::APPLICABILITY_ARRAY_SIZE * sizeof(size_t));
    hipMemset(d_applicability, 0, Sym::APPLICABILITY_ARRAY_SIZE * sizeof(size_t));
    mem_total += Sym::APPLICABILITY_ARRAY_SIZE * sizeof(size_t);

    size_t h_expression_count = expressions.size();
    size_t* d_expression_count;
    hipMalloc(&d_expression_count, sizeof(size_t));
    mem_total += sizeof(size_t);

    std::cout << "Allocated " << mem_total << " bytes (" << mem_total / 1024 / 1024 << "MiB)" << std::endl;

    std::cout << "Copying to GPU memory" << std::endl;

    hipMemcpy(d_expression_count, &h_expression_count, sizeof(size_t), hipMemcpyHostToDevice);
    for (size_t i = 0; i < expressions.size(); ++i) {
        hipMemcpy(d_expressions + Sym::EXPRESSION_MAX_SYMBOL_COUNT * i, expressions[i].data(),
                   expressions[i].size() * sizeof(Sym::Symbol), hipMemcpyHostToDevice);
    }

    std::cout << "Checking heuristics" << std::endl;

    Sym::check_for_known_integrals<<<BLOCK_COUNT, BLOCK_SIZE>>>(d_expressions, d_applicability,
                                                                d_expression_count);

    std::cout << "Calculating partial sum of applicability" << std::endl;

    thrust::inclusive_scan(thrust::device, d_applicability,
                           d_applicability + Sym::APPLICABILITY_ARRAY_SIZE, d_applicability);

    std::cout << "Applying heuristics" << std::endl;

    Sym::apply_known_integrals<<<BLOCK_COUNT, BLOCK_SIZE>>>(d_expressions, d_expressions_swap,
                                                            d_applicability, d_expression_count);
    std::swap(d_expressions, d_expressions_swap);
    hipMemcpy(d_expression_count, d_applicability + Sym::APPLICABILITY_ARRAY_SIZE - 1,
               sizeof(size_t), hipMemcpyDeviceToDevice);

    std::cout << "Copying results to host memory" << std::endl;

    std::vector<size_t> h_applicability(Sym::APPLICABILITY_ARRAY_SIZE);
    hipMemcpy(h_applicability.data(), d_applicability,
               Sym::APPLICABILITY_ARRAY_SIZE * sizeof(size_t), hipMemcpyDeviceToHost);

    std::vector<Sym::Symbol> h_results(Sym::EXPRESSION_ARRAY_SIZE);
    hipMemcpy(h_results.data(), d_expressions, Sym::EXPRESSION_ARRAY_SIZE * sizeof(Sym::Symbol),
               hipMemcpyDeviceToHost);
    hipMemcpy(&h_expression_count, d_expression_count, sizeof(size_t), hipMemcpyDeviceToHost);

    std::cout << "Applicability:" << std::endl;
    for (size_t i = 0; i < h_applicability.size(); ++i) {
        if (i % Sym::MAX_EXPRESSION_COUNT == 0 && i != 0) {
            std::cout << std::endl;
        }

        std::cout << h_applicability[i] << ", ";
    }
    std::cout << std::endl;

    std::cout << "Results: " << std::endl;
    for (size_t i = 0; i < h_expression_count; ++i) {
        std::cout << h_results[i * Sym::EXPRESSION_MAX_SYMBOL_COUNT].to_string() << std::endl;
    }

    std::cout << "Freeing GPU memory" << std::endl;
    hipFree(d_applicability);
    hipFree(d_expressions_swap);
    hipFree(d_expressions);
}

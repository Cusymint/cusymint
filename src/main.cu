#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstring>

#include <iostream>
#include <vector>

#include <thrust/execution_policy.h>
#include <thrust/scan.h>

#include "integrate.cuh"
#include "symbol.cuh"

int main() {
    std::cout << "Creating an expression" << std::endl;
    std::vector<std::vector<Sym::Symbol>> expressions = {Sym::cos(Sym::var()),
                                                         Sym::sin(Sym::cos(Sym::var())),
                                                         Sym::e() ^ Sym::var(),
                                                         Sym::var() ^ Sym::num(5),
                                                         Sym::var() ^ Sym::pi(),
                                                         Sym::var() ^ Sym::var(),
                                                         Sym::pi()};
    std::cout << "Expression created" << std::endl;

    for (size_t i = 0; i < expressions.size(); ++i) {
        std::cout << expressions[i][0].to_string() << std::endl;
    }

    std::cout << "Allocating GPU memory" << std::endl;

    std::vector<Sym::Symbol*> h_d_expressions(expressions.size());

    for (size_t i = 0; i < expressions.size(); ++i) {
        hipMalloc(&h_d_expressions[i], expressions[i].size() * sizeof(Sym::Symbol));
    }

    Sym::Symbol** d_expressions;
    hipMalloc(&d_expressions, expressions.size() * sizeof(Sym::Symbol*));

    size_t* d_applicability;
    hipMalloc(&d_applicability, Sym::APPLICABILITY_SIZE * sizeof(size_t));
    hipMemset(d_applicability, 0, Sym::APPLICABILITY_SIZE * sizeof(size_t));

    std::cout << "Allocated GPU memory" << std::endl;
    std::cout << "Copying to GPU memory" << std::endl;

    for (size_t i = 0; i < expressions.size(); ++i) {
        hipMemcpy(h_d_expressions[i], expressions[i].data(),
                   expressions[i].size() * sizeof(Sym::Symbol), hipMemcpyHostToDevice);
    }

    hipMemcpy(d_expressions, h_d_expressions.data(), h_d_expressions.size() * sizeof(Sym::Symbol*),
               hipMemcpyHostToDevice);

    std::cout << "Copied to GPU memory" << std::endl;
    std::cout << "Checking heuristics" << std::endl;

    Sym::check_heuristics_applicability<<<32, 1024>>>(d_expressions, d_applicability,
                                                      expressions.size());

    std::cout << "Checked heuristics" << std::endl;
    std::cout << "Calculating partial sum of applicability" << std::endl;

    thrust::inclusive_scan(thrust::device, d_applicability,
                           d_applicability + Sym::APPLICABILITY_SIZE, d_applicability);

    std::cout << "Calculated partial sum of applicability" << std::endl;
    std::cout << "Copying results to host memory" << std::endl;

    std::vector<size_t> h_applicability(expressions.size());
    h_applicability.resize(Sym::APPLICABILITY_SIZE);
    hipMemcpy(h_applicability.data(), d_applicability, Sym::APPLICABILITY_SIZE * sizeof(size_t),
               hipMemcpyDeviceToHost);

    std::cout << "Copied results to host memory" << std::endl;

    for (size_t i = 0; i < h_applicability.size(); ++i) {
        if (i % Sym::MAX_EXPRESSION_COUNT == 0) {
            std::cout << std::endl;
        }

        std::cout << h_applicability[i] << ", ";
    }

    std::cout << std::endl;

    std::cout << "Freeing GPU memory" << std::endl;
    for (size_t i = 0; i < expressions.size(); ++i) {
        hipFree(h_d_expressions[i]);
    }
    hipFree(d_expressions);
    std::cout << "Freed GPU memory" << std::endl;
}

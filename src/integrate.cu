#include "hip/hip_runtime.h"
#include "integrate.cuh"

#include "cuda_utils.cuh"

namespace Sym {
    __device__ ApplicabilityCheck known_integral_checks[] = {is_simple_variable_power,
                                                             is_variable_exponent, is_simple_sine,
                                                             is_simple_cosine, is_constant};

    __device__ IntegralTransform known_integral_applications[] = {
        integrate_simple_variable_power, integrate_variable_exponent, integrate_simple_sine,
        integrate_simple_cosine, integrate_constant};

    static_assert(sizeof(known_integral_applications) == sizeof(known_integral_checks),
                  "Different number of heuristics and applications defined");

    static_assert(sizeof(known_integral_checks) ==
                      sizeof(ApplicabilityCheck) * KNOWN_INTEGRAL_COUNT,
                  "HEURISTIC_CHECK_COUNT is not equal to number of heuristic checks");

    __device__ size_t dummy_heuristic_check(Symbol*) { return 0; }
    __device__ void dummy_heuristic_transform(Symbol*, Symbol*) {}

    __device__ ApplicabilityCheck heuristic_checks[] = {dummy_heuristic_check};

    __device__ IntegralTransform heuristic_applications[] = {dummy_heuristic_transform};

    static_assert(sizeof(heuristic_checks) == sizeof(heuristic_applications),
                  "Different number of heuristics and applications defined");

    static_assert(sizeof(heuristic_checks) == sizeof(ApplicabilityCheck) * HEURISTIC_CHECK_COUNT,
                  "HEURISTIC_CHECK_COUNT is not equal to number of heuristic checks");

    __device__ size_t is_simple_variable_power(Symbol* integral) {
        return integral[0].is(Type::Power) && integral[1].is(Type::Variable) &&
               (integral[2].is(Type::NumericConstant) &&
                    integral[2].numeric_constant.value != 0.0 ||
                integral[2].is(Type::KnownConstant) || integral[2].is(Type::UnknownConstant));
    }
    __device__ size_t is_variable_exponent(Symbol* integral) {
        return integral[0].is(Type::Power) && integral[1].is(Type::KnownConstant) &&
               integral[1].known_constant.value == KnownConstantValue::E &&
               integral[2].is(Type::Variable);
    }
    __device__ size_t is_simple_sine(Symbol* integral) {
        return integral[0].is(Type::Sine) && integral[1].is(Type::Variable);
    }

    __device__ size_t is_simple_cosine(Symbol* integral) {
        return integral[0].is(Type::Cosine) && integral[1].is(Type::Variable);
    }

    __device__ size_t is_constant(Symbol* integral) {
        return integral[0].is(Type::NumericConstant) || integral[0].is(Type::KnownConstant) ||
               integral[0].is(Type::UnknownConstant);
    }

    // TODO: Sometimes results in "too many resources requested for launch" error when block size is
    // 1024?
    __device__ void integrate_simple_variable_power(Symbol* integral, Symbol* destination) {
        size_t exponent_size = integral[2].unknown.total_size;

        destination[0].product = Product::create();
        destination[0].product.second_arg_offset = 5;
        destination[0].product.total_size = 8 + 2 * exponent_size;

        destination[1].reciprocal = Reciprocal::create();
        destination[1].reciprocal.total_size = 3 + exponent_size;

        destination[2].addition = Addition::create();
        destination[2].addition.total_size = 2 + exponent_size;
        destination[2].addition.second_arg_offset = 2;

        // copy exponent
        for (size_t i = 0; i < exponent_size; ++i) {
            destination[3 + i] = integral[2 + i];
        }

        destination[3 + exponent_size].numeric_constant = NumericConstant::create();
        destination[3 + exponent_size].numeric_constant.value = 1.0;

        destination[4 + exponent_size].power = Power::create();
        destination[4 + exponent_size].power.second_arg_offset = 2;
        destination[4 + exponent_size].power.total_size = 4 + exponent_size;

        destination[5 + exponent_size] = integral[1]; // copy variable

        destination[6 + exponent_size].addition = Addition::create();
        destination[6 + exponent_size].addition.second_arg_offset = 2;
        destination[6 + exponent_size].addition.total_size = 2 + exponent_size;

        // copy exponent
        for (size_t i = 0; i < exponent_size; ++i) {
            destination[7 + exponent_size + i] = integral[2 + i];
        }

        destination[7 + exponent_size * 2].numeric_constant = NumericConstant::create();
        destination[7 + exponent_size * 2].numeric_constant.value = 1.0;
    }

    __device__ void integrate_variable_exponent(Symbol* integral, Symbol* destination) {
        destination[0] = integral[0]; // power
        destination[1] = integral[1]; // e constant
        destination[2] = integral[2]; // variable
    }

    __device__ void integrate_simple_sine(Symbol* integral, Symbol* destination) {
        destination[0].negative = Negative::create();
        destination[0].negative.total_size = 3;

        destination[1].cosine = Cosine::create();
        destination[1].cosine.total_size = 2;

        destination[2] = integral[1]; // copy variable
    }

    __device__ void integrate_simple_cosine(Symbol* integral, Symbol* destination) {
        destination[0].sine = Sine::create();
        destination[0].sine.total_size = 2;

        destination[1] = integral[1]; // copy variable
    }

    __device__ void integrate_constant(Symbol* integral, Symbol* destination) {
        destination[0].product = Product::create();
        destination[0].product.total_size = 3;
        destination[0].product.second_arg_offset = 2;
        destination[1].variable = Variable::create();
        destination[2] = integral[0]; // copy constant
    }

    __device__ void check_applicability(Symbol* integrals, size_t* applicability,
                                        size_t* integral_count, ApplicabilityCheck* checks,
                                        size_t check_count) {
        size_t thread_count = Util::thread_count();
        size_t thread_idx = Util::thread_idx();

        size_t check_step = thread_count / TRANSFORM_GROUP_SIZE;

        for (size_t check_idx = thread_idx / TRANSFORM_GROUP_SIZE; check_idx < check_count;
             check_idx += check_step) {
            for (size_t expr_idx = thread_idx % TRANSFORM_GROUP_SIZE; expr_idx < *integral_count;
                 expr_idx += TRANSFORM_GROUP_SIZE) {
                Symbol* integral_pointer = integrals + expr_idx * INTEGRAL_MAX_SYMBOL_COUNT;
                size_t applicability_index = MAX_INTEGRAL_COUNT * check_idx + expr_idx;

                applicability[applicability_index] = checks[check_idx](integral_pointer);
            }
        }
    }

    __device__ void apply_transforms(Symbol* integrals, Symbol* destinations,
                                     size_t* applicability, size_t* integral_count,
                                     IntegralTransform* transforms, size_t transform_count) {
        size_t thread_count = Util::thread_count();
        size_t thread_idx = Util::thread_idx();

        size_t trans_step = thread_count / TRANSFORM_GROUP_SIZE;

        for (size_t trans_idx = thread_idx / TRANSFORM_GROUP_SIZE; trans_idx < transform_count;
             trans_idx += trans_step) {
            for (size_t expr_idx = thread_idx % TRANSFORM_GROUP_SIZE; expr_idx < *integral_count;
                 expr_idx += TRANSFORM_GROUP_SIZE) {
                Symbol* integral_pointer = integrals + expr_idx * INTEGRAL_MAX_SYMBOL_COUNT;
                size_t applicability_index = MAX_INTEGRAL_COUNT * trans_idx + expr_idx;

                if (applicability_index == 0 && applicability[applicability_index] != 0 ||
                    applicability_index != 0 && applicability[applicability_index - 1] !=
                                                    applicability[applicability_index]) {
                    size_t destination_offset =
                        INTEGRAL_MAX_SYMBOL_COUNT * (applicability[applicability_index] - 1);
                    Symbol* destination = destinations + destination_offset;

                    transforms[trans_idx](integral_pointer, destination);
                }
            }
        }
    }

    __global__ void check_for_known_integrals(Symbol* integrals, size_t* applicability,
                                              size_t* integral_count) {
        check_applicability(integrals, applicability, integral_count, known_integral_checks,
                            KNOWN_INTEGRAL_COUNT);
    }

    __global__ void apply_known_integrals(Symbol* integrals, Symbol* destinations,
                                          size_t* applicability, size_t* integral_count) {
        apply_transforms(integrals, destinations, applicability, integral_count,
                         known_integral_applications, KNOWN_INTEGRAL_COUNT);
    }

    __global__ void check_heuristics_applicability(Symbol* integrals, size_t* applicability,
                                                   size_t* integral_count) {
        check_applicability(integrals, applicability, integral_count, heuristic_checks,
                            HEURISTIC_CHECK_COUNT);
    }

    __global__ void apply_heuristics(Symbol* integrals, Symbol* destinations,
                                     size_t* applicability, size_t* integral_count) {
        apply_transforms(integrals, destinations, applicability, integral_count,
                         heuristic_applications, HEURISTIC_CHECK_COUNT);
    }
}

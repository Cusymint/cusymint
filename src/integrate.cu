#include "hip/hip_runtime.h"
#include "integrate.cuh"

namespace Sym {
    __device__ HeuristicCheck heuristic_checks[HEURISTIC_CHECK_COUNT] = {
        is_simple_variable_power, is_variable_exponent, is_simple_sine, is_simple_cosine, is_sum};

    __device__ bool is_simple_variable_power(Symbol* expression) {
        return expression[0].is(Type::Power) && expression[1].is(Type::Variable) &&
               (expression[2].is(Type::NumericConstant) &&
                    expression[2].numeric_constant.value != 0.0 ||
                expression[2].is(Type::KnownConstant) || expression[2].is(Type::UnknownConstant));
    }
    __device__ bool is_variable_exponent(Symbol* expression) {
        return expression[0].is(Type::Power) && expression[1].is(Type::KnownConstant) &&
               expression[1].known_constant.value == KnownConstantValue::E &&
               expression[2].is(Type::Variable);
    }
    __device__ bool is_simple_sine(Symbol* expression) {
        return expression[0].is(Type::Sine) && expression[1].is(Type::Variable);
    }

    __device__ bool is_simple_cosine(Symbol* expression) {
        return expression[0].is(Type::Cosine) && expression[1].is(Type::Variable);
    }

    __device__ bool is_sum(Symbol* expression) { return expression[0].is(Type::Addition); }

    __global__ void check_heuristics_applicability(Symbol** expressions, bool** applicability,
                                                   size_t expression_count) {
        size_t thread_count = gridDim.x * blockDim.x;
        size_t thread_idx = threadIdx.x + blockDim.x * blockIdx.x;

        for (size_t hrstc_idx = thread_idx / HEURISITC_GROUP_SIZE;
             hrstc_idx < HEURISTIC_CHECK_COUNT; hrstc_idx += thread_count / HEURISITC_GROUP_SIZE) {
            for (size_t expr_idx = thread_idx % HEURISITC_GROUP_SIZE; expr_idx < expression_count;
                 expr_idx += HEURISITC_GROUP_SIZE) {
                applicability[expr_idx][hrstc_idx] =
                    heuristic_checks[hrstc_idx](expressions[expr_idx]);
            }
        }
    }
} // namespace Sym
